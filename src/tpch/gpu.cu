#include "hip/hip_runtime.h"
#include <bits/stdint-intn.h>
#include <bits/stdint-uintn.h>
#include <cstddef>
#include <iostream>
#include <math.h>
#include <cassert>
#include <cstring>

#include "common.hpp"

struct group {
    uint64_t sum_qty;
    uint64_t sum_base_price;
    uint64_t sum_disc_price;
    uint64_t sum_charge;
    uint64_t avg_qty;
    uint64_t avg_price;
    uint64_t avg_disc;
    uint64_t count_order;
    char l_returnflag;
    char l_linestatus;
};

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}


__device__ void ht_insert(int32_t k)
{
}

__device__ group* createGroup() {
    group* ptr = (group*)malloc(sizeof(group));
    printf("Thread %d got pointer: %p\n", threadIdx.x, ptr);
    memset(ptr, 0, sizeof(group));
    return ptr;
}

/*
-- TPC-H Query 1

select
        l_returnflag,
        l_linestatus,
        sum(l_quantity) as sum_qty,
        sum(l_extendedprice) as sum_base_price,
        sum(l_extendedprice * (1 - l_discount)) as sum_disc_price,
        sum(l_extendedprice * (1 - l_discount) * (1 + l_tax)) as sum_charge,
        avg(l_quantity) as avg_qty,
        avg(l_extendedprice) as avg_price,
        avg(l_discount) as avg_disc,
        count(*) as count_order
from
        lineitem
where
        l_shipdate <= date '1998-12-01' - interval '90' day
group by
        l_returnflag,
        l_linestatus
order by
        l_returnflag,
        l_linestatus
*/

__managed__ group* globalHT[16];

__global__
void query_1_kernel(int n, char* l_returnflag, char* l_linestatus, int64_t* l_quantity, int64_t* l_extendedprice, int64_t* l_discount, int64_t* l_tax, uint32_t* l_shipdate)
{
    //constexpr auto threshold_date = to_julian_day(2, 9, 1998); // 1998-09-02
    uint32_t threshold_date = 2451059;

    __shared__ group* ht[16];
    for (int i = threadIdx.x; i < 16; i += blockDim.x) {
        ht[i] = nullptr;
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (l_shipdate[i] > threshold_date) continue;

        uint16_t h = static_cast<uint16_t>(l_returnflag[i]) << 8;
        h |= l_linestatus[i];
        h &= 0b0111;
        group* groupPtr = ht[h];
        if (groupPtr != nullptr) {
            if (groupPtr->l_returnflag != l_returnflag[i] || groupPtr->l_linestatus != l_linestatus[i]) {
                // TODO handle collisions
                __threadfence();
                printf("trap\n");
                asm("trap;");
            }
        } else {
            // create new group
            groupPtr = createGroup();
            groupPtr->l_returnflag = l_returnflag[i];
            groupPtr->l_linestatus = l_linestatus[i];

            // atomicCAS(int* address, int compare, int val);
            auto stored = atomicCAS((unsigned long long int*)&ht[h], 0ull, (unsigned long long int)groupPtr);
            if (stored != 0ull) {
                free(groupPtr);
                groupPtr = ht[h];
            }
        }
/*
        auto current_l_extendedprice = l_extendedprice[i];
        auto current_l_discount = l_discount[i];*/
        auto current_l_quantity = l_quantity[i];

        atomicAdd((unsigned long long int*)&groupPtr->sum_qty, (unsigned long long int)current_l_quantity);

        atomicAdd((unsigned long long int*)&groupPtr->count_order, 1ull);
    }

  //  __sync
  //  printf("done\n");

/*
struct group {
    uint64_t sum_qty;
    uint64_t sum_base_price;
    uint64_t sum_disc_price;
    uint64_t sum_charge;
    uint64_t avg_qty;
    uint64_t avg_price;
    uint64_t avg_disc;
    uint64_t count_order;
*/
    for (int i = threadIdx.x; i < 16; i += blockDim.x) {
        group* localGroup = ht[i];
        group* globalGroup = globalHT[i];

        if (globalGroup == nullptr) {
            auto stored = atomicCAS((unsigned long long int*)&globalHT[i], 0ull, (unsigned long long int)localGroup);
            if (stored != 0ull) {
                globalGroup = globalHT[i];
            }
        }

        if (localGroup != globalGroup) {
            atomicAdd((unsigned long long int*)&globalGroup->sum_qty, (unsigned long long int)localGroup->sum_qty);

            atomicAdd((unsigned long long int*)&globalGroup->count_order, (unsigned long long int)localGroup->count_order);
        }

    }


    printf("%d\n", index);
}

__global__ void mallocTest()
{
    char* ptr = (char*)malloc(123);
    printf("Thread %d got pointer: %p\n", threadIdx.x, ptr);
    free(ptr);
}

/*
struct lineitem_table_t {
    std::vector<uint32_t> l_orderkey;
    std::vector<uint32_t> l_partkey;
    std::vector<uint32_t> l_suppkey;
    std::vector<uint32_t> l_linenumber;
    std::vector<int64_t> l_quantity;
    std::vector<int64_t> l_extendedprice;
    std::vector<int64_t> l_discount;
    std::vector<int64_t> l_tax;
    std::vector<char> l_returnflag;
    std::vector<char> l_linestatus;
    std::vector<uint32_t> l_shipdate;
    std::vector<uint32_t> l_commitdate;
    std::vector<uint32_t> l_receiptdate;
    std::vector<std::array<char, 25>> l_shipinstruct;
    std::vector<std::array<char, 10>> l_shipmode;
    std::vector<std::string> l_comment;
};
*/

int main(int argc, char** argv) {
    assert(argc > 1);
    Database db;
    load_tables(db, argv[1]);


    const auto N = db.lineitem.l_commitdate.size();
    lineitem_table_mgd_t lineitem;

    size_t columnSize = N*sizeof(decltype(db.lineitem.l_orderkey)::value_type);
    hipMallocManaged(&lineitem.l_orderkey, columnSize);
    std::memcpy(lineitem.l_orderkey, db.lineitem.l_orderkey.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_partkey)::value_type);
    hipMallocManaged(&lineitem.l_partkey, columnSize);
    std::memcpy(lineitem.l_partkey, db.lineitem.l_partkey.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_suppkey)::value_type);
    hipMallocManaged(&lineitem.l_suppkey, columnSize);
    std::memcpy(lineitem.l_suppkey, db.lineitem.l_suppkey.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_linenumber)::value_type);
    hipMallocManaged(&lineitem.l_linenumber, columnSize);
    std::memcpy(lineitem.l_linenumber, db.lineitem.l_linenumber.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_quantity)::value_type);
    hipMallocManaged(&lineitem.l_quantity, columnSize);
    std::memcpy(lineitem.l_quantity, db.lineitem.l_quantity.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_extendedprice)::value_type);
    hipMallocManaged(&lineitem.l_extendedprice, columnSize);
    std::memcpy(lineitem.l_extendedprice, db.lineitem.l_extendedprice.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_discount)::value_type);
    hipMallocManaged(&lineitem.l_discount, columnSize);
    std::memcpy(lineitem.l_discount, db.lineitem.l_discount.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_tax)::value_type);
    hipMallocManaged(&lineitem.l_tax, columnSize);
    std::memcpy(lineitem.l_tax, db.lineitem.l_tax.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_returnflag)::value_type);
    hipMallocManaged(&lineitem.l_returnflag, columnSize);
    std::memcpy(lineitem.l_returnflag, db.lineitem.l_returnflag.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_linestatus)::value_type);
    hipMallocManaged(&lineitem.l_linestatus, columnSize);
    std::memcpy(lineitem.l_linestatus, db.lineitem.l_linestatus.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_shipdate)::value_type);
    hipMallocManaged(&lineitem.l_shipdate, columnSize);
    std::memcpy(lineitem.l_shipdate, db.lineitem.l_shipdate.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_commitdate)::value_type);
    hipMallocManaged(&lineitem.l_commitdate, columnSize);
    std::memcpy(lineitem.l_commitdate, db.lineitem.l_commitdate.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_receiptdate)::value_type);
    hipMallocManaged(&lineitem.l_receiptdate, columnSize);
    std::memcpy(lineitem.l_receiptdate, db.lineitem.l_receiptdate.data(), columnSize);
/*
    columnSize = N*sizeof(decltype(db.lineitem.l_shipinstruct)::value_type);
    hipMallocManaged(&lineitem.l_shipinstruct, columnSize);
    std::memcpy(lineitem.l_shipinstruct, db.lineitem.l_shipinstruct.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_shipmode)::value_type);
    hipMallocManaged(&lineitem.l_shipmode, columnSize);
    std::memcpy(lineitem.l_shipmode, db.lineitem.l_shipmode.data(), columnSize);

    columnSize = N*sizeof(decltype(db.lineitem.l_comment)::value_type);
    hipMallocManaged(&lineitem.l_comment, columnSize);
    std::memcpy(lineitem.l_comment, db.lineitem.l_comment.data(), columnSize);
*/

    // Set a heap size of 128 megabytes. Note that this must
    // be done before any kernel is launched.
    cudaThreadSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
#if 0
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;

    int N = 1<<20;
    float *x, *y;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
#endif

    std::memset(globalHT, 0, 16*sizeof(void*));

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    // char* l_returnflag, char* l_linestatus, int64_t* l_quantity, int64_t* l_extendedprice, int64_t* l_discount, int64_t* l_tax, uint32_t* l_shipdate
    query_1_kernel<<<numBlocks, blockSize>>>(N,
        lineitem.l_returnflag, lineitem.l_linestatus, lineitem.l_quantity, lineitem.l_extendedprice, lineitem.l_discount, lineitem.l_tax, lineitem.l_shipdate);

    hipDeviceSynchronize();
    for (unsigned i = 0; i < 16; i++) {
        if (globalHT[i] != nullptr) {
            printf("group %d - count: %lu\n", i, globalHT[i]->count_order);
        }
    }

    return 0;
}
