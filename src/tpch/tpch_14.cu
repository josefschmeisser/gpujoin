#include "hip/hip_runtime.h"
#include "common.hpp"

#include <algorithm>
#include <bits/stdint-uintn.h>
#include <cstddef>
#include <cstdio>
#include <device_atomic_functions.h>
#include <hip/driver_types.h>
#include <iostream>
#include <limits>
#include <math.h>
#include <cassert>
#include <cstring>
#include <chrono>


//#include <cub/block/block_load.cuh>
//#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>

//#include "thirdparty/cub_test/test_util.h"


#include "LinearProbingHashTable.cuh"
#include "btree.cuh"
#include "btree.cu"
#include "rs.cu"

using namespace hipcub;

using vector_copy_policy = vector_to_managed_array;
using rs_placement_policy = vector_to_managed_array;

static constexpr bool prefetch_index = false;
static constexpr bool sort_indexed_relation = true;
static constexpr int block_size = 128;
static int num_sms;

const uint32_t lower_shipdate = 2449962; // 1995-09-01
const uint32_t upper_shipdate = 2449992; // 1995-10-01

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__managed__ int tupleCount;

using device_ht_t = LinearProbingHashTable<uint32_t, size_t>::DeviceHandle;

__global__ void hj_build_kernel(size_t n, const part_table_plain_t* part, device_ht_t ht) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        ht.insert(part->p_partkey[i], i);
    }
}

__device__ int my_strcmp(const char *str_a, const char *str_b, unsigned len){
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) {
            done = 1;
        } else if (str_a[i] != str_b[i]) {
            match = i+1;
            if (((int)str_a[i] - (int)str_b[i]) < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__managed__ int64_t globalSum1 = 0;
__managed__ int64_t globalSum2 = 0;

#define FULL_MASK 0xffffffff

// see: https://stackoverflow.com/a/44337310
__forceinline__ __device__ unsigned lane_id() {
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__global__ void hj_probe_kernel(size_t n, const part_table_plain_t* __restrict__ part, const lineitem_table_plain_t* __restrict__ lineitem, device_ht_t ht) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        size_t part_tid;
        bool match = ht.lookup(lineitem->l_partkey[i], part_tid);
        // TODO refill
        if (match) {
            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}


struct btree_index {
    const btree::Node* tree_;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        auto tree = btree::construct(h_column, 0.7);
        if (prefetch_index) {
            btree::prefetchTree(tree, 0);
        }
        tree_ = tree;
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
        return btree::cuda::btree_lookup(tree_, key);
    //    return btree::cuda::btree_lookup_with_hints(tree_, key); // TODO
    }
};

struct radix_spline_index {
    rs::DeviceRadixSpline* d_rs_;
    const btree::key_t* d_column_;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        d_column_ = d_column;
        auto h_rs = rs::build_radix_spline(h_column);

        // copy radix spline
        const auto start = std::chrono::high_resolution_clock::now();
        d_rs_ = rs::copy_radix_spline<rs_placement_policy>(h_rs);
        const auto finish = std::chrono::high_resolution_clock::now();
        const auto duration = chrono::duration_cast<chrono::microseconds>(finish - start).count()/1000.;
        std::cout << "radixspline transfer time: " << duration << " ms\n";

        auto rrs __attribute__((unused)) = reinterpret_cast<const rs::RawRadixSpline*>(&h_rs);
        assert(h_column.size() == rrs->num_keys_);
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
        const unsigned estimate = rs::cuda::get_estimate(d_rs_, key);
        const unsigned begin = (estimate < d_rs_->max_error_) ? 0 : (estimate - d_rs_->max_error_);
        const unsigned end = (estimate + d_rs_->max_error_ + 2 > d_rs_->num_keys_) ? d_rs_->num_keys_ : (estimate + d_rs_->max_error_ + 2);

        const auto bound_size = end - begin;
        const unsigned pos = begin + rs::cuda::lower_bound(key, &d_column_[begin], bound_size, [] (const rs::rs_key_t& a, const rs::rs_key_t& b) -> int {
            return a < b;
        });
        return (pos < d_rs_->num_keys_) ? static_cast<btree::payload_t>(pos) : btree::invalidTid;
    }
};

struct lower_bound_index {
    struct device_data_t {
        const btree::key_t* d_column;
        const unsigned d_size;
    }* device_data;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        device_data_t tmp { d_column, static_cast<unsigned>(h_column.size()) };
        hipMalloc(&device_data, sizeof(device_data_t));
        hipMemcpy(device_data, &tmp, sizeof(device_data_t), hipMemcpyHostToDevice);
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
//        return btree::cuda::branchy_binary_search(key, device_data->d_column, device_data->d_size);
        return btree::cuda::branch_free_binary_search(key, device_data->d_column, device_data->d_size);
    }
};

using chosen_index_structure = radix_spline_index;// btree_index;

template<class IndexStructureType>
__global__ void ij_full_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const unsigned lineitem_size, const part_table_plain_t* __restrict__ part, IndexStructureType index_structure) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        auto payload = index_structure(lineitem->l_partkey[i]);
        if (payload != btree::invalidTid) {
            const size_t part_tid = reinterpret_cast<size_t>(payload);

            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}


template<class T>
__device__ T atomic_sub_safe(T* address, T val) {
    unsigned expected, update, old;
    old = *address;
    do {
        expected = old;
        update = (old - val < old) ? (old - val) : 0;
        old = atomicCAS(address, expected, update);
    } while (expected != old);
    return old;
}


template<class T>
__forceinline__ __device__ T round_up_pow2(T value) {
    return static_cast<T>(1) << (sizeof(T)*8 - __clz(value - 1));
}

template<
    int   BLOCK_THREADS,
    int   ITEMS_PER_THREAD,
    class IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_full_kernel_2(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
//    const part_table_plain_t* __restrict__ part,
    IndexStructureType index_structure)
{
/*
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;
*/

    enum {
        MAX_ITEMS_PER_WARP = ITEMS_PER_THREAD * 32,
        WARPS_PER_BLOCK = BLOCK_THREADS / 32,
        // the last summand ensures that each thread can write one more element during the last scan iteration
        BUFFER_SIZE = ITEMS_PER_THREAD*BLOCK_THREADS + BLOCK_THREADS
    };
    typedef BlockRadixSort<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    __shared__ uint32_t l_partkey_buffer[BUFFER_SIZE];
    __shared__ uint32_t lineitem_tid_buffer[BUFFER_SIZE];
    __shared__ uint32_t buffer_idx;

    __shared__ uint32_t fully_occupied_warps;
    __shared__ uint32_t exhausted_warps;
    __shared__ uint32_t items_per_warp[WARPS_PER_BLOCK];

// TODO
//    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

    union {
        struct {
            uint32_t l_partkey;
            uint32_t lineitem_tid;
        } join_pair;
        uint64_t raw;
    } join_pairs[ITEMS_PER_THREAD];

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;

const unsigned tile_size_raw = (lineitem_size + BLOCK_THREADS - 1)/gridDim.x;
    const unsigned tile_size = round_up_pow2((lineitem_size + BLOCK_THREADS - 1) / gridDim.x);
    if (warp_id == 0 && lane_id == 0) { printf("lineitem_size: %d gridDim.x: %d tile_size: %d tile_size_raw: %d\n", lineitem_size, gridDim.x, tile_size, tile_size_raw); }
    unsigned tid = blockIdx.x * tile_size + threadIdx.x;
    const unsigned tid_limit = min(tid + tile_size, lineitem_size);
//return;

    // initialize shared variables
    if (warp_id == 0 && lane_id == 0) {
        buffer_idx = 0;
        fully_occupied_warps = 0;
        exhausted_warps = 0;
    }

    while (exhausted_warps < WARPS_PER_BLOCK) {
        __syncthreads(); // ensure that all shared variables are initialized


if (lane_id == 0) { printf("items_per_warp[%d]: %d\n", warp_id, items_per_warp[warp_id]); }

        uint16_t local_idx = 0;
        uint32_t underfull_lanes = FULL_MASK; // lanes that have less than ITEMS_PER_THREAD items in their registers
        uint32_t unexhausted_lanes = FULL_MASK; // lanes which can still fetch new tuples

        //unsigned tid = threadIdx.x + thread_offset;
        //unsigned thread_offset = lane_id;

        while (unexhausted_lanes && underfull_lanes && items_per_warp[warp_id] < MAX_ITEMS_PER_WARP) {

            if (lane_id == 0) { printf("warp: %d first tid: %d\n", warp_id, tid); }

            //if (lane_id == 0) { printf("underfull_lanes: 0x%.8X\n", underfull_lanes); }

            if (lane_id == 0) { printf("items_per_warp[%d]: %d\n", warp_id, items_per_warp[warp_id]); }

            int active = tid < tid_limit;

            // filter predicate
            if (active) {
                active = lineitem->l_shipdate[tid] >= lower_shipdate && lineitem->l_shipdate[tid] < upper_shipdate;
            }

            // fetch attributes
            uint32_t l_partkey;
            if (active) {
                l_partkey = lineitem->l_partkey[tid];
            }

            // negotiate buffer target positions among all threads in this warp
            unsigned mask = __ballot_sync(FULL_MASK, local_idx >= ITEMS_PER_THREAD);
            unsigned right = __funnelshift_l(0xffffffff, 0, lane_id);
            unsigned offset = __popc(mask & right);
            unsigned dest_idx = 0;
            if (active && mask && lane_id == 0) {
                dest_idx = atomicAdd(&buffer_idx, __popc(mask));
                printf("warp: %d dest_idx: %d\n", warp_id, dest_idx);
            }
            dest_idx = __shfl_sync(FULL_MASK, dest_idx, 0);

            // matrialize attributes
            if (active && local_idx >= ITEMS_PER_THREAD) {
                // buffer items
                lineitem_tid_buffer[dest_idx] = tid;
                l_partkey_buffer[dest_idx] = l_partkey;
            } else if (active) {
                // store items in registers
                auto& p = join_pairs[local_idx++].join_pair;
                p.lineitem_tid = tid;
                p.l_partkey = l_partkey;
            }

            underfull_lanes = __ballot_sync(FULL_MASK, local_idx < ITEMS_PER_THREAD); // FIXME
            unexhausted_lanes = __ballot_sync(FULL_MASK, tid < tid_limit);
if (lane_id == 0) { printf("underfull_lanes: 0x%.8X\n", underfull_lanes); }

            if (unexhausted_lanes == 0 && lane_id == 0) {
                //atomicInc(&exhausted_warps, std::numeric_limits<decltype(exhausted_warps)>::max());
                atomicInc(&exhausted_warps, UINT_MAX);
            }

            auto active_lanes = __ballot_sync(FULL_MASK, active);
            if (lane_id == 0) {
                printf("active_lanes: 0x%.8X\n", active_lanes);
                atomicAdd(&items_per_warp[warp_id], __popc(active_lanes));
            }

            tid += BLOCK_THREADS; // each tile is organized as a consecutive succession of its corresponding block

            __syncwarp();
        }
        if (lane_id == 0) { printf("warp: %d unexhausted_lanes: 0x%.8X\n", warp_id, unexhausted_lanes); }

        __syncthreads(); // wait until all threads have gathered enough elements

        // determine the number of items required to fully populate this warp
        const unsigned required = ITEMS_PER_THREAD - local_idx;
printf("warp: %d lane: %d required: %d\n", warp_id, lane_id, required);
        unsigned ideal_refill_cnt = required;
/*
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);

*/
        if (underfull_lanes) {
            #pragma unroll
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                ideal_refill_cnt += __shfl_down_sync(FULL_MASK, ideal_refill_cnt, offset);
            }
        }
//__syncwarp();
if (lane_id == 0) { printf("warp: %d ideal_refill_cnt: %d buffer_idx: %d\n", warp_id, ideal_refill_cnt, buffer_idx); }
        // distribute buffered items among the threads in this warp
        if (ideal_refill_cnt > 0) {
            int available_cnt = 0;
            if (lane_id == 0) {
                auto old = atomic_sub_safe(&buffer_idx, ideal_refill_cnt);
                available_cnt = (old > ideal_refill_cnt) ? ideal_refill_cnt : old;
            }

            //T __shfl_sync(unsigned mask, T var, int srcLane, int width=warpSize);
            available_cnt = __shfl_sync(FULL_MASK, available_cnt, 0);

            int prefix_sum = required;
//printf("warp: %d lane: %d required: %d\n", warp_id, lane_id, prefix_sum);
            // calculate the inclusive prefix sum among all threads in this warp
            #pragma unroll
            for (int offset = 1; offset <= 32; offset <<= 1) {
                auto value = __shfl_up_sync(FULL_MASK, prefix_sum, offset);
                prefix_sum += (lane_id >= offset) ? value : 0;
            }
            // calculate the exclusive prefix sum
            prefix_sum -= required;
printf("warp: %d lane: %d prefix_sum: %d\n", warp_id, lane_id, prefix_sum);

            // refill registers with buffered elements
            const auto limit = prefix_sum + required;
            for (; prefix_sum < limit; ++prefix_sum) {
                auto& p = join_pairs[local_idx++].join_pair;
                p.lineitem_tid = lineitem_tid_buffer[prefix_sum];
                p.l_partkey = l_partkey_buffer[prefix_sum];
            }

            ideal_refill_cnt -= available_cnt;
        }
if (lane_id == 0) printf("ideal_refill_cnt: %d\n", ideal_refill_cnt);

        if (ideal_refill_cnt == 0 && lane_id == 0) {
            //atomicInc(&fully_occupied_warps, std::numeric_limits<decltype(fully_occupied_warps)>::max());
            atomicInc(&fully_occupied_warps, UINT_MAX);
        }

        __syncthreads(); // wait until all threads have tried to fill their registers
if (lane_id == 0) printf("fully_occupied_warps: %d\n", fully_occupied_warps);

        if (fully_occupied_warps == WARPS_PER_BLOCK) {
            if (warp_id == 0 && lane_id == 0) printf("=== sorting... ===\n");
            /* TODO
            BlockRadixSortT(temp_storage).SortBlockedToStriped(join_pairs, 20, 32); // TODO
            */
        }

if (warp_id == 0 && lane_id == 0) { printf("start sorting\n"); }
/*
        for (unsigned i = 0; i < actual_items; ++i) {
            btree::payload_t payload = btree::invalidTid;
            payload = index_structure();
        }
*/
        // reset state
        __syncthreads(); // wait until each wrap is done
        if (lane_id == 0) {
            fully_occupied_warps = 0;
            items_per_warp[warp_id] = 0;
        }

if (warp_id == 0 && lane_id == 0) { printf("exhausted_warps: %d\n", exhausted_warps); }

    }
}


struct JoinEntry {
    unsigned lineitem_tid;
    unsigned part_tid;
};
__device__ unsigned output_index = 0;

/*
// source: https://developer.nvidia.com/blog/using-cuda-warp-level-primitives/
// increment the value at ptr by 1 and return the old value
__device__ int atomicAggInc(int *ptr) {
    int mask = __match_any_sync(__activemask(), (unsigned long long)ptr);
    int leader = __ffs(mask) - 1;    // select a leader
    int res;
    if(lane_id() == leader)                  // leader does the update
        res = atomicAdd(ptr, __popc(mask));
    res = __shfl_sync(mask, res, leader);    // get leader’s old value
    return res + __popc(mask & ((1 << lane_id()) - 1)); //compute old value
}*/

template<class IndexStructureType>
__global__ void ij_lookup_kernel(const lineitem_table_plain_t* __restrict__ lineitem, unsigned lineitem_size, const IndexStructureType index_structure, JoinEntry* __restrict__ join_entries) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size + 31; i += stride) {
        btree::payload_t payload = btree::invalidTid;
        if (i < lineitem_size &&
            lineitem->l_shipdate[i] >= lower_shipdate &&
            lineitem->l_shipdate[i] < upper_shipdate) {
            payload = index_structure(lineitem->l_partkey[i]);
        }

        int match = payload != btree::invalidTid;
        unsigned mask = __ballot_sync(FULL_MASK, match);
        unsigned my_lane = lane_id();
        unsigned right = __funnelshift_l(0xffffffff, 0, my_lane);
//        printf("right %u\n", right);
        unsigned offset = __popc(mask & right);

        unsigned base = 0;
        int leader = __ffs(mask) - 1;
        if (my_lane == leader) {
            base = atomicAdd(&output_index, __popc(mask));
        }
        base = __shfl_sync(FULL_MASK, base, leader);

        if (match) {
//            printf("lane %u store to: %u\n", my_lane, base + offset);
            auto& join_entry = join_entries[base + offset];
            join_entry.lineitem_tid = i;
            join_entry.part_tid = payload;
        }
    }
}

__global__ void ij_join_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const part_table_plain_t* __restrict__ part, const JoinEntry* __restrict__ join_entries, size_t n) {
    int64_t sum1 = 0;
    int64_t sum2 = 0;
    const char* prefix = "PROMO";

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        const auto lineitem_tid = join_entries[i].lineitem_tid;
        const auto part_tid = join_entries[i].part_tid;

        const auto extendedprice = lineitem->l_extendedprice[lineitem_tid];
        const auto discount = lineitem->l_discount[lineitem_tid];
        const auto summand = extendedprice * (100 - discount);
        sum2 += summand;

        const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//        printf("type: %s\n", type);
        if (my_strcmp(type, prefix, 5) == 0) {
            sum1 += summand;
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    //__reduce_add_sync() requires compute capability 8
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}

template<class IndexType>
struct helper {
    IndexType index_structure;

    unsigned lineitem_size;
    lineitem_table_plain_t* lineitem_device;
    std::unique_ptr<lineitem_table_plain_t> lineitem_device_ptrs;

    unsigned part_size;
    part_table_plain_t* part_device;
    std::unique_ptr<part_table_plain_t> part_device_ptrs;

    void load_database(const std::string& path) {
        Database db;
        load_tables(db, path);
        if (sort_indexed_relation) {
            printf("sorting part relation...\n");
            sort_relation(db.part);
        }
        lineitem_size = db.lineitem.l_orderkey.size();
        part_size = db.part.p_partkey.size();

        {
            const auto start = std::chrono::high_resolution_clock::now();
            //auto [lineitem_device, lineitem_device_ptrs] = copy_relation<vector_copy_policy>(db.lineitem);
            std::tie(lineitem_device, lineitem_device_ptrs) = copy_relation<vector_copy_policy>(db.lineitem);
            //auto [part_device, part_device_ptrs] = copy_relation<vector_copy_policy>(db.part);
            std::tie(part_device, part_device_ptrs) = copy_relation<vector_copy_policy>(db.part);
            const auto finish = std::chrono::high_resolution_clock::now();
            const auto d = chrono::duration_cast<chrono::milliseconds>(finish - start).count();
            std::cout << "transfer time: " << d << " ms\n";
        }

#ifndef USE_HJ
        index_structure.construct(db.part.p_partkey, part_device_ptrs->p_partkey);
#endif
    }

#ifdef USE_HJ
    void run_hj() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        LinearProbingHashTable<uint32_t, size_t> ht(part_size);
        int num_blocks = (part_size + block_size - 1) / block_size;
        hj_build_kernel<<<num_blocks, block_size>>>(part_size, part_device, ht.deviceHandle);

        //num_blocks = 32*num_sms;
        num_blocks = (lineitem_size + block_size - 1) / block_size;
        hj_probe_kernel<<<num_blocks, block_size>>>(lineitem_size, part_device, lineitem_device, ht.deviceHandle);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }
#endif

    void run_ij() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_full_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, part_device, index_structure);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

#if 0
    void run_two_phase_ij() {
        JoinEntry* join_entries;
        hipMalloc(&join_entries, sizeof(JoinEntry)*lineitem_size);

        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (part_size + block_size - 1) / block_size;
        ij_lookup_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, index_structure, join_entries);
        hipDeviceSynchronize();

        decltype(output_index) matches;
        hipError_t error = hipMemcpyFromSymbol(&matches, HIP_SYMBOL(output_index), sizeof(matches), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        //printf("join matches: %u\n", matches);

        num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries, matches);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();local_idx
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }
#else
/*
template<
    int   BLOCK_THREADS,
    int   ITEMS_PER_THREAD,
    class IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_full_kernel_2(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
//    const part_table_plain_t* __restrict__ part,
    IndexStructureType index_structure)
*/


    void run_two_phase_ij() {

        enum { BLOCK_THREADS = 64, ITEMS_PER_THREAD = 4 };

        JoinEntry* join_entries;
        hipMalloc(&join_entries, sizeof(JoinEntry)*lineitem_size);

        const auto start1 = std::chrono::high_resolution_clock::now();

        int num_blocks = 1;// TODO
        ij_full_kernel_2<BLOCK_THREADS, ITEMS_PER_THREAD, IndexType><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, 1024*2048, index_structure);
        hipDeviceSynchronize();

        const auto d1 = chrono::duration_cast<chrono::microseconds>(std::chrono::high_resolution_clock::now() - start1).count()/1000.;
        std::cout << "kernel time: " << d1 << " ms\n";



        decltype(output_index) matches;
        hipError_t error = hipMemcpyFromSymbol(&matches, HIP_SYMBOL(output_index), sizeof(matches), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        //printf("join matches: %u\n", matches);

        num_blocks = (lineitem_size + block_size - 1) / block_size;

        const auto start2 = std::chrono::high_resolution_clock::now();
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries, matches);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - start2).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }


#endif
};

template<class IndexType>
void load_and_run_ij(const std::string& path, bool as_full_pipline_breaker) {
    helper<IndexType> h;
    h.load_database(path);
    if (as_full_pipline_breaker) {
        printf("full pipline breaker\n");
        h.run_two_phase_ij();
    } else {
        h.run_ij();
    }
}

int main(int argc, char** argv) {
    using namespace std;

    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);// devId);

#ifdef USE_HJ
    if (argc != 2) {
        printf("%s <tpch dataset path>\n", argv[0]);
        return 0;
    }

    helper<lower_bound_index> h;
    h.load_database(argv[1]);
    h.run_hj();
#else
    if (argc < 3) {
        printf("%s <tpch dataset path> <index type: {0: btree, 1: radixspline, 2: lowerbound> <1: full pipline breaker>\n", argv[0]);
        return 0;
    }
    enum IndexType : unsigned { btree, radixspline, lowerbound } index_type { static_cast<IndexType>(std::stoi(argv[2])) };
    bool full_pipline_breaker = (argc < 4) ? false : std::stoi(argv[3]) != 0;

    switch (index_type) {
        case IndexType::btree: {
            printf("using btree\n");
            load_and_run_ij<btree_index>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::radixspline: {
            printf("using radixspline\n");
            load_and_run_ij<radix_spline_index>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::lowerbound: {
            printf("using lower bound search\n");
            load_and_run_ij<lower_bound_index>(argv[1], full_pipline_breaker);
            break;
        }
        default:
            std::cerr << "unknown index type: " << index_type << std::endl;
            return 0;
    }
#endif

/*
    printf("sum1: %lu\n", globalSum1);
    printf("sum2: %lu\n", globalSum2);
*/
    const int64_t result = 100*(globalSum1*1'000)/(globalSum2/1'000);
    printf("%ld.%ld\n", result/1'000'000, result%1'000'000);

    return 0;
}
