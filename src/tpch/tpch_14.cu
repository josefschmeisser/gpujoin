#include "hip/hip_runtime.h"
#include "common.hpp"

#include <algorithm>
#include <cstddef>
#include <cstdio>
#include <device_atomic_functions.h>
#include <hip/driver_types.h>
#include <iostream>
#include <limits>
#include <math.h>
#include <cassert>
#include <cstring>
#include <chrono>

#include "LinearProbingHashTable.cuh"
#include "btree.cuh"
#include "btree.cu"
#include "rs.cu"

using vector_copy_policy = vector_to_managed_array;
using rs_placement_policy = vector_to_managed_array;

static constexpr bool prefetch_index = false;
static constexpr bool sort_indexed_relation = true;
static constexpr int block_size = 128;
static int num_sms;

const uint32_t lower_shipdate = 2449962; // 1995-09-01
const uint32_t upper_shipdate = 2449992; // 1995-10-01

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__managed__ int tupleCount;

using device_ht_t = LinearProbingHashTable<uint32_t, size_t>::DeviceHandle;

__global__ void hj_build_kernel(size_t n, const part_table_plain_t* part, device_ht_t ht) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        ht.insert(part->p_partkey[i], i);
    }
}

__device__ int my_strcmp(const char *str_a, const char *str_b, unsigned len){
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) {
            done = 1;
        } else if (str_a[i] != str_b[i]) {
            match = i+1;
            if (((int)str_a[i] - (int)str_b[i]) < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__managed__ int64_t globalSum1 = 0;
__managed__ int64_t globalSum2 = 0;

#define FULL_MASK 0xffffffff

// see: https://stackoverflow.com/a/44337310
__forceinline__ __device__ unsigned lane_id() {
    unsigned ret; 
    asm volatile ("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}

__global__ void hj_probe_kernel(size_t n, const part_table_plain_t* __restrict__ part, const lineitem_table_plain_t* __restrict__ lineitem, device_ht_t ht) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        size_t part_tid;
        bool match = ht.lookup(lineitem->l_partkey[i], part_tid);
        // TODO refill
        if (match) {
            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}


struct btree_index {
    const btree::Node* tree_;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        auto tree = btree::construct(h_column, 0.7);
        if (prefetch_index) {
            btree::prefetchTree(tree, 0);
        }
        tree_ = tree;
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
        return btree::cuda::btree_lookup(tree_, key);
    //    return btree::cuda::btree_lookup_with_hints(tree_, key); // TODO
    }
};

struct radix_spline_index {
    rs::DeviceRadixSpline* d_rs_;
    const btree::key_t* d_column_;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        d_column_ = d_column;
        auto h_rs = rs::build_radix_spline(h_column);
        d_rs_ = rs::copy_radix_spline<rs_placement_policy>(h_rs);
        auto rrs __attribute__((unused)) = reinterpret_cast<const rs::RawRadixSpline*>(&h_rs);
        assert(h_column.size() == rrs->num_keys_);
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
        const unsigned estimate = rs::cuda::get_estimate(d_rs_, key);
        const unsigned begin = (estimate < d_rs_->max_error_) ? 0 : (estimate - d_rs_->max_error_);
        const unsigned end = (estimate + d_rs_->max_error_ + 2 > d_rs_->num_keys_) ? d_rs_->num_keys_ : (estimate + d_rs_->max_error_ + 2);

        const auto bound_size = end - begin;
        const unsigned pos = begin + rs::cuda::lower_bound(key, &d_column_[begin], bound_size, [] (const rs::rs_key_t& a, const rs::rs_key_t& b) -> int {
            return a < b;
        });
        return (pos < d_rs_->num_keys_) ? static_cast<btree::payload_t>(pos) : btree::invalidTid;
    }
};

struct lower_bound_index {
    struct device_data_t {
        const btree::key_t* d_column;
        const unsigned d_size;
    }* device_data;

    __host__ void construct(const std::vector<btree::key_t>& h_column, const btree::key_t* d_column) {
        device_data_t tmp { d_column, static_cast<unsigned>(h_column.size()) };
        hipMalloc(&device_data, sizeof(device_data_t));
        hipMemcpy(device_data, &tmp, sizeof(device_data_t), hipMemcpyHostToDevice);
    }

    __device__ __forceinline__ btree::payload_t operator() (const btree::key_t key) const {
//        return btree::cuda::branchy_binary_search(key, device_data->d_column, device_data->d_size);
        return btree::cuda::branch_free_binary_search(key, device_data->d_column, device_data->d_size);
    }
};

using chosen_index_structure = radix_spline_index;// btree_index;// radix_spline_index;


template<class IndexStructureType>
__global__ void ij_full_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const unsigned lineitem_size, const part_table_plain_t* __restrict__ part, IndexStructureType index_structure) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        auto payload = index_structure(lineitem->l_partkey[i]);
        if (payload != btree::invalidTid) {
            const size_t part_tid = reinterpret_cast<size_t>(payload);

            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}


struct JoinEntry {
    unsigned lineitem_tid;
    unsigned part_tid;
};
__device__ unsigned output_index = 0;

/*
// source: https://developer.nvidia.com/blog/using-cuda-warp-level-primitives/
// increment the value at ptr by 1 and return the old value
__device__ int atomicAggInc(int *ptr) {
    int mask = __match_any_sync(__activemask(), (unsigned long long)ptr);
    int leader = __ffs(mask) - 1;    // select a leader
    int res;
    if(lane_id() == leader)                  // leader does the update
        res = atomicAdd(ptr, __popc(mask));
    res = __shfl_sync(mask, res, leader);    // get leader’s old value
    return res + __popc(mask & ((1 << lane_id()) - 1)); //compute old value
}*/

template<class IndexStructureType>
__global__ void ij_lookup_kernel(const lineitem_table_plain_t* __restrict__ lineitem, unsigned lineitem_size, const IndexStructureType index_structure, JoinEntry* __restrict__ join_entries) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size + 31; i += stride) {
        btree::payload_t payload = btree::invalidTid;
        if (i < lineitem_size &&
            lineitem->l_shipdate[i] >= lower_shipdate &&
            lineitem->l_shipdate[i] < upper_shipdate) {
            payload = index_structure(lineitem->l_partkey[i]);
        }

        int match = payload != btree::invalidTid;
        unsigned mask = __ballot_sync(FULL_MASK, match);
        unsigned my_lane = lane_id();
        unsigned right = __funnelshift_l(0xffffffff, 0, my_lane);
//        printf("right %u\n", right);
        unsigned offset = __popc(mask & right);

        unsigned base = 0;
        int leader = __ffs(mask) - 1;
        if (my_lane == leader) {
            base = atomicAdd(&output_index, __popc(mask));
        }
        base = __shfl_sync(FULL_MASK, base, leader);

        if (match) {
//            printf("lane %u store to: %u\n", my_lane, base + offset);
            auto& join_entry = join_entries[base + offset];
            join_entry.lineitem_tid = i;
            join_entry.part_tid = payload;
        }
    }
}

__global__ void ij_join_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const part_table_plain_t* __restrict__ part, const JoinEntry* __restrict__ join_entries, size_t n) {
    int64_t sum1 = 0;
    int64_t sum2 = 0;
    const char* prefix = "PROMO";

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        const auto lineitem_tid = join_entries[i].lineitem_tid;
        const auto part_tid = join_entries[i].part_tid;

        const auto extendedprice = lineitem->l_extendedprice[lineitem_tid];
        const auto discount = lineitem->l_discount[lineitem_tid];
        const auto summand = extendedprice * (100 - discount);
        sum2 += summand;

        const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//        printf("type: %s\n", type);
        if (my_strcmp(type, prefix, 5) == 0) {
            sum1 += summand;
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}

template<class IndexType>
struct helper {
    IndexType index_structure;

    unsigned lineitem_size;
    lineitem_table_plain_t* lineitem_device;
    std::unique_ptr<lineitem_table_plain_t> lineitem_device_ptrs;

    unsigned part_size;
    part_table_plain_t* part_device;
    std::unique_ptr<part_table_plain_t> part_device_ptrs;

    void load_database(const std::string& path) {
        Database db;
        load_tables(db, path);
        if (sort_indexed_relation) {
            printf("sorting part relation...\n");
            sort_relation(db.part);
        }
        lineitem_size = db.lineitem.l_orderkey.size();
        part_size = db.part.p_partkey.size();

        {
            const auto start = std::chrono::high_resolution_clock::now();
            //auto [lineitem_device, lineitem_device_ptrs] = copy_relation<vector_copy_policy>(db.lineitem);
            std::tie(lineitem_device, lineitem_device_ptrs) = copy_relation<vector_copy_policy>(db.lineitem);
            //auto [part_device, part_device_ptrs] = copy_relation<vector_copy_policy>(db.part);
            std::tie(part_device, part_device_ptrs) = copy_relation<vector_copy_policy>(db.part);
            const auto finish = std::chrono::high_resolution_clock::now();
            const auto d = chrono::duration_cast<chrono::milliseconds>(finish - start).count();
            std::cout << "transfer time: " << d << " ms\n";
        }

#ifndef USE_HJ
        index_structure.construct(db.part.p_partkey, part_device_ptrs->p_partkey);
#endif
    }

#ifdef USE_HJ
    void run_hj() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        LinearProbingHashTable<uint32_t, size_t> ht(part_size);
        int num_blocks = (part_size + block_size - 1) / block_size;
        hj_build_kernel<<<num_blocks, block_size>>>(part_size, part_device, ht.deviceHandle);

        //num_blocks = 32*num_sms;
        num_blocks = (lineitem_size + block_size - 1) / block_size;
        hj_probe_kernel<<<num_blocks, block_size>>>(lineitem_size, part_device, lineitem_device, ht.deviceHandle);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }
#endif

    void run_ij() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_full_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, part_device, index_structure);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

    void run_two_phase_ij() {
        JoinEntry* join_entries;
        hipMalloc(&join_entries, sizeof(JoinEntry)*lineitem_size);

        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (part_size + block_size - 1) / block_size;
        ij_lookup_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, index_structure, join_entries);
        hipDeviceSynchronize();

        decltype(output_index) matches;
        hipError_t error = hipMemcpyFromSymbol(&matches, HIP_SYMBOL(output_index), sizeof(matches), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        //printf("join matches: %u\n", matches);

        num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries, matches);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }
};

template<class IndexType>
void load_and_run_ij(const std::string& path, bool as_full_pipline_breaker) {
    helper<IndexType> h;
    h.load_database(path);
    if (as_full_pipline_breaker) {
        printf("full pipline breaker\n");
        h.run_two_phase_ij();
    } else {
        h.run_ij();
    }
}

int main(int argc, char** argv) {
    using namespace std;

    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);// devId);

#ifdef USE_HJ
    if (argc != 2) {
        printf("%s <tpch dataset path>\n", argv[0]);
        return 0;
    }

    helper<lower_bound_index> h;
    h.load_database(argv[1]);
    h.run_hj();
#else
    if (argc < 3) {
        printf("%s <tpch dataset path> <index type: {0: btree, 1: radixspline, 2: lowerbound> <1: full pipline breaker>\n", argv[0]);
        return 0;
    }
    enum IndexType : unsigned { btree, radixspline, lowerbound } index_type { static_cast<IndexType>(std::stoi(argv[2])) };
    bool full_pipline_breaker = (argc < 4) ? false : std::stoi(argv[3]) != 0;

    switch (index_type) {
        case IndexType::btree: {
            printf("using btree\n");
            load_and_run_ij<btree_index>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::radixspline: {
            printf("using radixspline\n");
            load_and_run_ij<radix_spline_index>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::lowerbound: {
            printf("using lower bound search\n");
            load_and_run_ij<lower_bound_index>(argv[1], full_pipline_breaker);
            break;
        }
        default:
            std::cerr << "unknown index type: " << index_type << std::endl;
            return 0;
    }
#endif

/*
    printf("sum1: %lu\n", globalSum1);
    printf("sum2: %lu\n", globalSum2);
*/
    const int64_t result = 100*(globalSum1*1'000)/(globalSum2/1'000);
    printf("%ld.%ld\n", result/1'000'000, result%1'000'000);

    return 0;
}
