#include "hip/hip_runtime.h"
#include "common.hpp"

#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <device_atomic_functions.h>
#include <hip/driver_types.h>
#include <iostream>
#include <iomanip>
#include <limits>
#include <cassert>
#include <cstring>
#include <chrono>
#include <unordered_map>

#include <cub/block/block_radix_sort.cuh>

#include "cuda_utils.cuh"
#include "LinearProbingHashTable.cuh"
#include "cuda_allocator.hpp"
#include "numa_allocator.hpp"
#include "mmap_allocator.hpp"
#include "indexes.cuh"
#include "device_array.hpp"

#define MEASURE_CYCLES
#define SKIP_SORT

using namespace hipcub;

//using vector_copy_policy = vector_to_managed_array; // TODO remove

using indexed_t = std::remove_pointer_t<decltype(lineitem_table_plain_t::l_partkey)>;
using payload_t = uint32_t;

// host allocator
//template<class T> using host_allocator = mmap_allocator<T, huge_2mb, 1>;
template<class T> using host_allocator = std::allocator<T>;
//template<class T> using host_allocator = cuda_allocator<T, cuda_allocation_type::zero_copy>;
//template<class T> using host_allocator = mmap_allocator<T, huge_2mb, 0>;

// device allocators
template<class T> using device_index_allocator = cuda_allocator<T, cuda_allocation_type::zero_copy>;
template<class T> using device_table_allocator = cuda_allocator<T, cuda_allocation_type::zero_copy>;
//template<class T> using device_index_allocator = mmap_allocator<T, huge_2mb, 0>;
//template<class T> using device_table_allocator = mmap_allocator<T, huge_2mb, 0>;


static constexpr bool prefetch_index __attribute__((unused)) = false;
static constexpr bool sort_indexed_relation = true;
static constexpr int block_size = 128;
static int num_sms;

static const uint32_t lower_shipdate = 2449962; // 1995-09-01
static const uint32_t upper_shipdate = 2449992; // 1995-10-01
static const uint32_t invalid_tid __attribute__((unused)) = std::numeric_limits<uint32_t>::max();

__device__ unsigned int count = 0;
__managed__ int tupleCount;

using device_ht_t = LinearProbingHashTable<uint32_t, size_t>::DeviceHandle;

__global__ void hj_build_kernel(size_t n, const part_table_plain_t* part, device_ht_t ht) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        ht.insert(part->p_partkey[i], i);
    }
}

__device__ int my_strcmp(const char *str_a, const char *str_b, unsigned len){
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) {
            done = 1;
        } else if (str_a[i] != str_b[i]) {
            match = i+1;
            if (((int)str_a[i] - (int)str_b[i]) < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__managed__ int64_t globalSum1 = 0;
__managed__ int64_t globalSum2 = 0;

__global__ void hj_probe_kernel(size_t n, const part_table_plain_t* __restrict__ part, const lineitem_table_plain_t* __restrict__ lineitem, device_ht_t ht) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        size_t part_tid;
        bool match = ht.lookup(lineitem->l_partkey[i], part_tid);
        // TODO refill
        if (match) {
            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}

template<class IndexStructureType>
__global__ void ij_full_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const unsigned lineitem_size, const part_table_plain_t* __restrict__ part, IndexStructureType index_structure) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        auto payload = index_structure.lookup(lineitem->l_partkey[i]);
        if (payload != invalid_tid) {
            const auto part_tid = reinterpret_cast<unsigned>(payload);

            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//            printf("type: %s\n", type);
            if (my_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}

struct JoinEntry {
    unsigned lineitem_tid;
    unsigned part_tid;
};
__device__ unsigned output_index = 0;

/*
// source: https://developer.nvidia.com/blog/using-cuda-warp-level-primitives/
// increment the value at ptr by 1 and return the old value
__device__ int atomicAggInc(int *ptr) {
    int mask = __match_any_sync(__activemask(), (unsigned long long)ptr);
    int leader = __ffs(mask) - 1;    // select a leader
    int res;
    if(lane_id() == leader)                  // leader does the update
        res = atomicAdd(ptr, __popc(mask));
    res = __shfl_sync(mask, res, leader);    // get leader’s old value
    return res + __popc(mask & ((1 << lane_id()) - 1)); //compute old value
}*/

template<class IndexStructureType>
__global__ void ij_lookup_kernel(const lineitem_table_plain_t* __restrict__ lineitem, unsigned lineitem_size, const IndexStructureType index_structure, JoinEntry* __restrict__ join_entries) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < lineitem_size + 31; i += stride) {
        payload_t payload = invalid_tid;
        if (i < lineitem_size &&
            lineitem->l_shipdate[i] >= lower_shipdate &&
            lineitem->l_shipdate[i] < upper_shipdate) {
            payload = index_structure.lookup(lineitem->l_partkey[i]);
        }

        int match = payload != invalid_tid;
        unsigned mask = __ballot_sync(FULL_MASK, match);
        unsigned my_lane = lane_id();
        unsigned right = __funnelshift_l(0xffffffff, 0, my_lane);
//        printf("right %u\n", right);
        unsigned offset = __popc(mask & right);

        unsigned base = 0;
        int leader = __ffs(mask) - 1;
        if (my_lane == leader) {
            base = atomicAdd(&output_index, __popc(mask));
        }
        base = __shfl_sync(FULL_MASK, base, leader);

        if (match) {
//            printf("lane %u store to: %u\n", my_lane, base + offset);
            auto& join_entry = join_entries[base + offset];
            join_entry.lineitem_tid = i;
            join_entry.part_tid = payload;
        }
    }
}

__device__ __forceinline__ unsigned testfun() { return 0; }

template<
    int   BLOCK_THREADS,
    class IndexStructureType >
__global__ void ij_lookup_kernel_4(const lineitem_table_plain_t* __restrict__ lineitem, unsigned lineitem_size, const IndexStructureType index_structure, JoinEntry* __restrict__ join_entries) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    const unsigned my_warp = threadIdx.x / 32;
    const unsigned my_lane = lane_id();
    const uint32_t right_mask = __funnelshift_l(0xffffffff, 0, my_lane);

    __shared__ uint32_t l_partkey_buffer[BLOCK_THREADS];
    __shared__ uint32_t lineitem_buffer_pos[BLOCK_THREADS];

    //__shared__ uint32_t buffer_idx;
    unsigned buffer_cnt = 0; // number of buffered items in this warp
    const unsigned buffer_start = 32*my_warp;

    // attributes
    uint32_t l_shipdate;
    uint32_t l_partkey;

    unsigned lineitem_tid;

    uint32_t unfinished_lanes = __ballot_sync(FULL_MASK, index < lineitem_size);
    while (unfinished_lanes || buffer_cnt > 0) {
        bool active = index < lineitem_size;
        if (active) {
            l_shipdate = lineitem->l_shipdate[index];
            l_partkey = lineitem->l_partkey[index];
            lineitem_tid = index;
        }

        if (active) {
            active = (l_shipdate >= lower_shipdate && l_shipdate < upper_shipdate);
        }

        const auto active_mask = __ballot_sync(FULL_MASK, active);
        auto active_cnt = __popc(active_mask);

        const unsigned threshold_cnt = (unfinished_lanes == 0) ? 0 : 25;
        while (buffer_cnt + active_cnt > threshold_cnt) {

            if (active_cnt < 25 && buffer_cnt > 0) {
                // refill
                const unsigned offset = __popc((~active_mask) & right_mask);

                const unsigned refill_cnt = min(buffer_cnt, 32 - active_cnt);

                if (!active && offset < buffer_cnt) {
                    const unsigned buffer_idx = buffer_start + buffer_cnt - offset - 1;
                    l_partkey = l_partkey_buffer[buffer_idx];
                    lineitem_tid = lineitem_buffer_pos[buffer_idx];
                    active = true;
                }

                buffer_cnt -= refill_cnt;
            }

            // next operator
            payload_t payload = index_structure.cooperative_lookup(active, l_partkey);

            const int match = payload != invalid_tid;
            const uint32_t mask = __ballot_sync(FULL_MASK, active && match);
            const unsigned offset = __popc(mask & right_mask);

            unsigned base = 0;
            if (my_lane == 0 && mask) {
                base = atomicAdd(&output_index, __popc(mask));
            }
            base = __shfl_sync(FULL_MASK, base, 0);

            if (active && match) {
                auto& join_entry = join_entries[base + offset];
                join_entry.lineitem_tid = lineitem_tid;
                join_entry.part_tid = payload;
            }

            active = false;
            active_cnt = 0;
        }

        if (active_cnt > 0) {
            // fill buffer
            const unsigned offset = __popc(active_mask & right_mask);
            if (active) {
                const unsigned buffer_idx = buffer_start + buffer_cnt + offset;
                l_partkey_buffer[buffer_idx] = l_partkey;
                lineitem_buffer_pos[buffer_idx] = lineitem_tid;
            }
            __syncwarp();

            buffer_cnt += active_cnt;
        }

        index += stride;
        unfinished_lanes = __ballot_sync(FULL_MASK, index < lineitem_size);
    }
}

template<
    int   BLOCK_THREADS,
    int   ITEMS_PER_THREAD,
    class IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_lookup_kernel_2(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
    IndexStructureType index_structure,
    JoinEntry* __restrict__ join_entries)
{
    enum {
        MAX_ITEMS_PER_WARP = ITEMS_PER_THREAD * 32,
        WARPS_PER_BLOCK = BLOCK_THREADS / 32,
        // the last summand ensures that each thread can write one more element during the last scan iteration
        BUFFER_SIZE = ITEMS_PER_THREAD*BLOCK_THREADS + BLOCK_THREADS,
        BUFFER_SOFT_LIMIT = ITEMS_PER_THREAD*BLOCK_THREADS
    };
    typedef BlockRadixSort<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    __shared__ uint32_t l_partkey_buffer[BUFFER_SIZE];
    __shared__ uint32_t lineitem_buffer_pos[BUFFER_SIZE];
    __shared__ uint32_t buffer_idx;

    __shared__ uint32_t fully_occupied_warps;
    __shared__ uint32_t exhausted_warps;

    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

    /*
    union {
        struct {
            uint32_t l_partkey;
            uint32_t lineitem_tid;
        } join_pair;
        uint64_t raw;
    } join_pairs[ITEMS_PER_THREAD];
*/
    union {
        uint64_t join_pairs_raw[ITEMS_PER_THREAD];
        struct {
            uint32_t l_partkey;
            uint32_t lineitem_tid;
        } join_pairs[ITEMS_PER_THREAD];
    };

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;

    const unsigned tile_size = min(lineitem_size, (lineitem_size + gridDim.x - 1) / gridDim.x);
    unsigned tid = blockIdx.x*tile_size; // first tid where the first thread of a block starts scanning
    const unsigned tid_limit = min(tid + tile_size, lineitem_size); // marks the end of each tile
    tid += threadIdx.x; // each thread starts at it's correponding offset

    // initialize shared variables
    if (warp_id == 0 && lane_id == 0) {
        buffer_idx = 0;
        fully_occupied_warps = 0;
        exhausted_warps = 0;
    }
    __syncthreads(); // ensure that all shared variables are initialized

    uint32_t unexhausted_lanes = FULL_MASK; // lanes which can still fetch new tuples

    while (exhausted_warps < WARPS_PER_BLOCK || buffer_idx > 0) {
        uint16_t local_idx = 0; // current size of the thread local array
        uint32_t underfull_lanes = FULL_MASK; // lanes that have less than ITEMS_PER_THREAD items in their registers (has to be reset after each iteration)

        while (unexhausted_lanes && underfull_lanes && buffer_idx < BUFFER_SOFT_LIMIT) {
            int active = tid < tid_limit;

            // TODO vectorize loads

            // filter predicate
            if (active) {
                active = lineitem->l_shipdate[tid] >= lower_shipdate && lineitem->l_shipdate[tid] < upper_shipdate;
            }

            // fetch attributes
            uint32_t l_partkey;
            if (active) {
                l_partkey = lineitem->l_partkey[tid];
            }

            // negotiate buffer target positions among all threads in this warp
            const uint32_t overflow_lanes = __ballot_sync(FULL_MASK, active && local_idx >= ITEMS_PER_THREAD);
            const uint32_t right = __funnelshift_l(FULL_MASK, 0, lane_id);
            uint32_t dest_idx = 0;
            if (overflow_lanes != 0 && lane_id == 0) {
                dest_idx = atomicAdd(&buffer_idx, __popc(overflow_lanes));
            }
            const uint32_t lane_offset = __popc(overflow_lanes & right);
            dest_idx = lane_offset + __shfl_sync(FULL_MASK, dest_idx, 0);

            // matrialize attributes
            if (active && local_idx >= ITEMS_PER_THREAD) {
                // buffer items
                lineitem_buffer_pos[dest_idx] = tid;
                l_partkey_buffer[dest_idx] = l_partkey;
            } else if (active) {
                // store items in registers
                auto& p = join_pairs[local_idx++];
                p.lineitem_tid = tid;
                p.l_partkey = l_partkey;
            }

            underfull_lanes = __ballot_sync(FULL_MASK, local_idx < ITEMS_PER_THREAD);
            unexhausted_lanes = __ballot_sync(FULL_MASK, tid < tid_limit);

            if (unexhausted_lanes == 0 && lane_id == 0) {
                atomicInc(&exhausted_warps, UINT_MAX);
            }

            tid += BLOCK_THREADS; // each tile is organized as a consecutive succession of its corresponding block
        }

        __syncthreads(); // wait until all threads have gathered enough elements

        // determine the number of items required to fully populate this lane
        const unsigned required = ITEMS_PER_THREAD - local_idx;
        int refill_cnt = 0;
        unsigned ideal_refill_cnt = required;

        // compute the number of required items across all lanes
        if (underfull_lanes) {
            #pragma unroll
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                ideal_refill_cnt += __shfl_down_sync(FULL_MASK, ideal_refill_cnt, offset);
            }
        }

        // distribute buffered items among the threads in this warp
        if (ideal_refill_cnt > 0) {
            uint32_t refill_idx_start;
            if (lane_id == 0) {
                const auto old = atomic_sub_safe(&buffer_idx, ideal_refill_cnt);
                refill_cnt = (old > ideal_refill_cnt) ? ideal_refill_cnt : old;
                refill_idx_start = old - refill_cnt;
            }

            refill_cnt = __shfl_sync(FULL_MASK, refill_cnt, 0);
            refill_idx_start = __shfl_sync(FULL_MASK, refill_idx_start, 0);

            int prefix_sum = required;
            // calculate the inclusive prefix sum among all threads in this warp
            #pragma unroll
            for (int offset = 1; offset < 32; offset <<= 1) {
                auto value = __shfl_up_sync(FULL_MASK, prefix_sum, offset);
                prefix_sum += (lane_id >= offset) ? value : 0;
            }
            // calculate the exclusive prefix sum
            prefix_sum -= required;

            // refill registers with buffered elements
            const auto limit = min(prefix_sum + required, refill_cnt);
            for (; prefix_sum < limit; ++prefix_sum) {
                auto& p = join_pairs[local_idx++];
                p.lineitem_tid = lineitem_buffer_pos[refill_idx_start + prefix_sum];
                p.l_partkey = l_partkey_buffer[refill_idx_start + prefix_sum];
            }

            ideal_refill_cnt -= refill_cnt;
        }

        if (ideal_refill_cnt == 0 && lane_id == 0) {
            atomicInc(&fully_occupied_warps, UINT_MAX);
        }

        __syncthreads(); // wait until all threads have tried to fill their registers

        if (fully_occupied_warps == WARPS_PER_BLOCK) {/*
            if (warp_id == 0 && lane_id == 0) printf("=== sorting... ===\n");
            assert(join_pairs[0].l_partkey == (join_pairs_raw[0] & FULL_MASK));
*/

uint64_t* arr = nullptr;
typedef uint64_t items_t[ITEMS_PER_THREAD];
items_t& test = (items_t&)arr;

            BlockRadixSortT(temp_storage).SortBlockedToStriped(test, 8, 21); // TODO

        }

        unsigned output_base = 0;
        const auto count = MAX_ITEMS_PER_WARP - ideal_refill_cnt;
        if (lane_id == 0) {
            output_base = atomicAdd(&output_index, count);
        }
        output_base = __shfl_sync(FULL_MASK, output_base, 0);

        int lane_dst_idx_prefix_sum = local_idx;
        // calculate the inclusive prefix sum among all threads in this warp
        #pragma unroll
        for (int offset = 1; offset < 32; offset <<= 1) {
            auto value = __shfl_up_sync(FULL_MASK, lane_dst_idx_prefix_sum, offset);
            lane_dst_idx_prefix_sum += (lane_id >= offset) ? value : 0;
        }
        lane_dst_idx_prefix_sum -= local_idx;
// FIXME warp excution order is not deterministic
        uint32_t active_lanes = __ballot_sync(FULL_MASK, local_idx > 0);
        for (unsigned i = 0; active_lanes != 0; ++i) {
            bool active = i < local_idx;
            auto& p = join_pairs[i];
            const auto tid = index_structure.cooperative_lookup(active, p.l_partkey);

            if (active) {
                assert(tid != invalid_tid);
                auto& join_entry = join_entries[output_base + lane_dst_idx_prefix_sum++];
                join_entry.lineitem_tid = p.lineitem_tid;
                join_entry.part_tid = tid;
            }
            active_lanes = __ballot_sync(FULL_MASK, active);
        }

        // reset state
        __syncthreads(); // wait until each wrap is done
        if (lane_id == 0) {
            fully_occupied_warps = 0;
        }
    }
}

#if 0
template<
    unsigned BLOCK_THREADS,
    unsigned ITEMS_PER_THREAD,
    class    IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_lookup_kernel_3(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
    const IndexStructureType index_structure,
    JoinEntry* __restrict__ join_entries)
{
    enum { ITEMS_PER_ITERATION = BLOCK_THREADS*ITEMS_PER_THREAD };

    // shared memory entry type
    union join_pair_t {
        struct { // TODO high low
            uint32_t l_partkey;
            uint32_t lineitem_tid;
        };
        uint64_t raw;
    };

    typedef hipcub::BlockRadixSort<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    __shared__ union TempStorage {
        // Allocate shared memory for BlockLoad
//        typename BlockLoad::TempStorage load;

        typename BlockRadixSortT::TempStorage sort;
    } temp_storage;

//    __shared__ uint64_t buffer[ITEMS_PER_ITERATION];
    __shared__ join_pair_t buffer[ITEMS_PER_ITERATION];
    __shared__ uint32_t buffer_pos;

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;

    const unsigned tile_size = min(lineitem_size, (lineitem_size + gridDim.x - 1) / gridDim.x);
    unsigned tid_begin = blockIdx.x * tile_size; // first tid where scanning starts at each new iteration
    const unsigned tid_limit = ; // TODO
//if (lane_id == 0) printf("warp: %d tile_size: %d\n", warp_id, tile_size);

    const unsigned iteration_count = (tile_size + ITEMS_PER_ITERATION - 1) / ITEMS_PER_ITERATION;

    using key_value_array_t = uint64_t[ITEMS_PER_THREAD];
    uint64_t* thread_data_raw = &buffer[threadIdx.x*ITEMS_PER_THREAD];
    key_value_array_t& thread_data = reinterpret_cast<key_value_array_t&>(*thread_data_raw);

    for (unsigned i = 0; i < iteration_count; ++i) {
        // reset shared memory variables
        if (lane_id == 0) {
            buffer_pos = 0;
        }

//if (lane_id == 0) printf("warp: %d iteration: %d first tid: %d\n", warp_id, i, tid);

        unsigned valid_items = min(ITEMS_PER_ITERATION, lineitem_size - tid_begin);
//if (lane_id == 0) printf("warp: %d valid_items: %d\n", warp_id, valid_items);

        #pragma unroll
        for (unsigned j = 0; j < ITEMS_PER_THREAD; ++j) {
            // sort_buffer::produce
            payload_t payload = invalid_tid;
            unsigned lineitem_tid = tid_begin + threadIdx.x*ITEMS_PER_THREAD + j;
            if (lineitem_tid < lineitem_size &&
                lineitem->l_shipdate[lineitem_tid] >= lower_shipdate &&
                lineitem->l_shipdate[lineitem_tid] < upper_shipdate)
            {
//                payload = index_structure(lineitem->l_partkey[i]);
                auto& join_pair = buffer[threadIdx.x*ITEMS_PER_THREAD + j];
                join_pair.lineitem_tid = lineitem_tid;
                join_pair.l_partkey = lineitem->l_partkey[lineitem_tid];
                assert(join_pair.raw & 0xffffffff == lineitem_tid); // TODO
            }
        }

        __syncthreads();

#if 1
        // we only perform the sort step when the buffer is completely filled
        if (valid_items == ITEMS_PER_ITERATION) {
//if (lane_id == 0) printf("warp: %d iteration: %d - sorting... ===\n", warp_id, i);
            BlockRadixSortT(temp_storage.sort).Sort(thread_data, 4, 24); // TODO
             __syncthreads();
        }
#endif

        // empty buffer
        unsigned old;
        do {
            if (lane_id == 0) {
                old = atomic_add_sat(&buffer_pos, 32u, valid_items);
            }
            old = __shfl_sync(FULL_MASK, old, 0);
            unsigned actual_count = min(valid_items - old, 32);
//if (lane_id == 0) printf("warp: %d iteration: %d - actual_count: %u\n", warp_id, i, actual_count);

            if (actual_count == 0) break;

            bool active = lane_id < actual_count;

            uint32_t assoc_tid = 0;
            key_t element = 0xffffffff;
            if (active) {
                assoc_tid = buffer[old + lane_id] >> 32;
                element = buffer[old + lane_id] & 0xffffffff;
//printf("warp: %d lane: %d - tid: %u element: %u\n", warp_id, lane_id, assoc_tid, element);
            }

            const auto tid_b = index_structure.cooperative_lookup(active, element);
            if (active) {
//printf("warp: %d lane: %d - tid_b: %u\n", warp_id, lane_id, tid_b);
//                tids[assoc_tid] = tid_b;
                // TODO
            }

//printf("warp: %d lane: $d - element: %u\n", warp_id, lane_id, );

        } while (true);//actual_count == 32);


        tid_begin += valid_items;
    }
}
#endif







__managed__ unsigned debug_cnt = 0;

template<
    unsigned BLOCK_THREADS,
    unsigned ITEMS_PER_THREAD,
    class    IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_lookup_kernel_3(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
    const IndexStructureType index_structure,
    JoinEntry* __restrict__ join_entries)
{
    enum {
        ITEMS_PER_WARP = ITEMS_PER_THREAD * 32, // soft upper limit
        ITEMS_PER_BLOCK = BLOCK_THREADS*ITEMS_PER_THREAD,
        WARPS_PER_BLOCK = BLOCK_THREADS / 32,
        // the last summand ensures that each thread can write one more element during the last scan iteration
//        BUFFER_SIZE = ITEMS_PER_THREAD*BLOCK_THREADS + BLOCK_THREADS,
        BUFFER_SIZE = BLOCK_THREADS*(ITEMS_PER_THREAD + 1)
//        BUFFER_SOFT_LIMIT = ITEMS_PER_THREAD*BLOCK_THREADS
    };

    // shared memory entry type
    union join_pair_t {
        struct { // TODO high low
            uint32_t l_partkey;
            uint32_t lineitem_tid;
        };
        uint64_t raw;
    };

    using BlockRadixSortT = hipcub::BlockRadixSort<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>;
    using key_value_array_t = uint64_t[ITEMS_PER_THREAD];

    __shared__ join_pair_t buffer[BUFFER_SIZE];
    __shared__ int buffer_idx;

    __shared__ uint32_t fully_occupied_warps;
    __shared__ uint32_t exhausted_warps;

    __shared__ union {
   //     uint16_t histogram[32]; // counts the respective msb
        typename BlockRadixSortT::TempStorage temp_storage;
    } temp_union;


    const float percentile = 0.9; // TODO
    const float r = 0.05;
    float moving_avg = -1.;
    float moving_seq_avg = -1.;
    float moving_percentile = -1.;

uint32_t max_partkey = 0;

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int max_reuse = warp_id*ITEMS_PER_WARP;
    const uint32_t right_mask = __funnelshift_l(FULL_MASK, 0, lane_id);

    const unsigned tile_size = min(lineitem_size, (lineitem_size + gridDim.x - 1) / gridDim.x);
    unsigned tid = blockIdx.x*tile_size; // first tid where the first thread of a block starts scanning
    const unsigned tid_limit = min(tid + tile_size, lineitem_size); // marks the end of each tile
    tid += threadIdx.x; // each thread starts at it's correponding offset

//if (lane_id == 0 && warp_id == 0) printf("lineitem_size: %u, gridDim.x: %u, tile_size: %u\n", lineitem_size, gridDim.x, tile_size);

    // initialize shared variables
    if (warp_id == 0 && lane_id == 0) {
        buffer_idx = 0;
        fully_occupied_warps = 0;
        exhausted_warps = 0;
    }
    __syncthreads(); // ensure that all shared variables are initialized

    uint32_t unexhausted_lanes = FULL_MASK; // lanes which can still fetch new tuples

    while (exhausted_warps < WARPS_PER_BLOCK || buffer_idx > 0) {
        // number of items stored in the buffer by this warp
        int warp_items = min(ITEMS_PER_WARP, max(0, buffer_idx - max_reuse));

        while (unexhausted_lanes && warp_items < ITEMS_PER_WARP) {
            int active = tid < tid_limit;

            // TODO vectorize loads

            // filter predicate
            if (active) {
                active = lineitem->l_shipdate[tid] >= lower_shipdate && lineitem->l_shipdate[tid] < upper_shipdate;
            }

            // fetch attributes
            uint32_t l_partkey;
            if (active) {
                l_partkey = lineitem->l_partkey[tid];
            }

            // negotiate buffer target positions among all threads in this warp
            const uint32_t active_mask = __ballot_sync(FULL_MASK, active);
            const auto item_cnt = __popc(active_mask);
            warp_items += item_cnt;
            uint32_t dest_idx = 0;
            if (lane_id == 0) {
                dest_idx = atomicAdd(&buffer_idx, item_cnt);
 //atomicAdd(&debug_cnt, item_cnt);
            }
            dest_idx = __shfl_sync(FULL_MASK, dest_idx, 0); // propagate the first buffer target index
            dest_idx += __popc(active_mask & right_mask); // add each participating thread's offset

            // matrialize attributes
            if (active) {
                auto& join_pair = buffer[dest_idx];
                join_pair.l_partkey = l_partkey;
                join_pair.lineitem_tid = tid;
//                printf("tid: %u\n", tid);
                /*
                printf("raw: %p rtid: %u rkey: %u tid1: %p key1: %p tid2: %u key2: %u\n", tid, l_partkey, (void*)join_pair.raw, (void*)(join_pair.raw & 0xffffffff),
                        (void*)(join_pair.raw >> 32), (unsigned)(join_pair.raw & 0xffffffff), (unsigned)(join_pair.raw >> 32));
                        */
//                printf("raw: %p tid: %u key: %u\n", (void*)join_pair.raw, (unsigned)(join_pair.raw & 0xffffffff), (unsigned)(join_pair.raw >> 32));
//                __threadfence();
/*
                assert((join_pair.raw & 0xffffffff) == l_partkey); // TODO
                assert((join_pair.raw >> 32) == tid); // TODO
*/

#if 0
                // update moving percentile
                if (moving_percentile < 0.) {
                    // initialize
                    moving_avg = l_partkey;
                    moving_seq_avg = l_partkey*l_partkey;
                    moving_percentile = l_partkey;
                } else {
                    moving_avg = r*l_partkey + (1. - r)*moving_avg;
                    auto current_var = moving_avg - l_partkey;
                    current_var *= current_var;
                    moving_seq_avg = r*current_var + (1. - r)*moving_seq_avg;

                    if (moving_percentile > l_partkey) {
                        moving_percentile -= sqrtf(moving_seq_avg)*r/percentile;
                    } else if (moving_percentile < l_partkey) {
                        moving_percentile += sqrtf(moving_seq_avg)*r/(1. - percentile);
                    }
                }
#endif
                max_partkey = (l_partkey > max_partkey) ? l_partkey : max_partkey;
            }

/*
// compute moving percentile
unsigned g = __popc(__ballot_sync(FULL_MASK, active && l_partkey > moving_percentile));
unsigned l = __popc(__ballot_sync(FULL_MASK, active && l_partkey < moving_percentile));

// end
*/

            unexhausted_lanes = __ballot_sync(FULL_MASK, tid < tid_limit);
            if (unexhausted_lanes == 0 && lane_id == 0) {
                atomicInc(&exhausted_warps, UINT_MAX);
            }

            tid += BLOCK_THREADS; // each tile is organized as a consecutive succession of elements belonging to the current thread block
        }

        if (lane_id == 0 && warp_items >= ITEMS_PER_WARP) {
            atomicInc(&fully_occupied_warps, UINT_MAX);
        }

        __syncthreads(); // wait until all threads have gathered enough elements

//if (lane_id == 0) printf("moving_percentile: %f avg: %f max_partkey: %u diff %f\n", moving_percentile, moving_avg, max_partkey, static_cast<float>(max_partkey)-moving_percentile);

#if 1
        if (fully_occupied_warps == WARPS_PER_BLOCK) {
//if (warp_id == 0 && lane_id == 0) printf("=== sorting... ===\n");

            const unsigned first_offset = max(0, static_cast<int>(buffer_idx) - ITEMS_PER_BLOCK);
            uint64_t* thread_data_raw = reinterpret_cast<uint64_t*>(&buffer[threadIdx.x*ITEMS_PER_THREAD + first_offset]);
            key_value_array_t& thread_data = reinterpret_cast<key_value_array_t&>(*thread_data_raw);

            BlockRadixSortT(temp_union.temp_storage).SortDescending(thread_data, 4, 22); // TODO
             __syncthreads();
        }
#endif


#if 1
        // empty buffer
        for (unsigned i = 0u; i < ITEMS_PER_THREAD; ++i) {
            unsigned old;
            if (lane_id == 0) {
//                old = atomic_add_sat(&buffer_pos, 32u, valid_items);
                // T atomic_sub_safe(T* address, T val)
                old = atomic_sub_safe(&buffer_idx, 32);
            }
            old = __shfl_sync(FULL_MASK, old, 0);
            const auto acquired_cnt = min(old, 32);
            const auto first_pos = old - acquired_cnt;
//if (lane_id == 0) printf("warp: %d iteration: %d - actual_count: %u\n", warp_id, i, actual_count);
//if (lane_id == 0) atomicAdd(&debug_cnt, acquired_cnt);

            if (acquired_cnt == 0u) break;

            bool active = lane_id < acquired_cnt;

            uint32_t assoc_tid = 0u;
            key_t element;
            if (active) {
/*
                const auto my_pos = first_pos + 31u - lane_id;
                assoc_tid = buffer[my_pos] >> 32;
                element = buffer[my_pos] & 0xffffffff;
*/
//                const auto& join_pair = buffer[first_pos + 31u - lane_id];
                const auto& join_pair = buffer[first_pos + acquired_cnt - 1 - lane_id]; // TODO check
                assoc_tid = join_pair.lineitem_tid;
                element = join_pair.l_partkey;
//printf("warp: %d lane: %d - tid: %u element: %u\n", warp_id, lane_id, assoc_tid, element);
            }

            payload_t tid_b = index_structure.cooperative_lookup(active, element);

            active = active && (tid_b != invalid_tid);

            // negotiate output buffer target positions
            const uint32_t active_mask = __ballot_sync(FULL_MASK, active);
            const auto item_cnt = __popc(active_mask);
//assert(item_cnt == acquired_cnt);
            uint32_t dest_idx = 0;
            if (lane_id == 0) {
                dest_idx = atomicAdd(&output_index, item_cnt);
            }
            dest_idx = __shfl_sync(FULL_MASK, dest_idx, 0); // propagate the first buffer target index
            dest_idx += __popc(active_mask & right_mask); // add each's participating thread's offset

            // write entry into ouput buffer
            if (active) {
//printf("warp: %d lane: %d - tid_b: %u\n", warp_id, lane_id, tid_b);
                auto& join_entry = join_entries[dest_idx];
                join_entry.lineitem_tid = assoc_tid;
                join_entry.part_tid = tid_b;
            }

//printf("warp: %d lane: $d - element: %u\n", warp_id, lane_id, );

        }
#else
        // discard elements
        if (lane_id == 0) buffer_idx = 0;
#endif

        // prepare next iteration
        if (lane_id == 0) {
            fully_occupied_warps = 0;
        }

        // reset moving percentile
        moving_percentile = -1.;

        __syncthreads();
    }
}




__managed__ unsigned long long lookup_cycles = 0;
__managed__ unsigned long long scan_cycles = 0;
__managed__ unsigned long long sync_cycles = 0;
__managed__ unsigned long long sort_cycles = 0;

template<
    unsigned BLOCK_THREADS,
    unsigned ITEMS_PER_THREAD,
    class    IndexStructureType >
__launch_bounds__ (BLOCK_THREADS)
__global__ void ij_full_kernel_2(
    const lineitem_table_plain_t* __restrict__ lineitem,
    const unsigned lineitem_size,
    const part_table_plain_t* __restrict__ part,
    const unsigned part_size,
    const IndexStructureType index_structure,
    int64_t* __restrict__ l_extendedprice_buffer,
    int64_t* __restrict__ l_discount_buffer
    )
{
    enum {
        ITEMS_PER_WARP = ITEMS_PER_THREAD * 32, // soft upper limit
        ITEMS_PER_BLOCK = BLOCK_THREADS*ITEMS_PER_THREAD,
        WARPS_PER_BLOCK = BLOCK_THREADS / 32,
        // the last summand ensures that each thread can write one more element during the last scan iteration
        BUFFER_SIZE = BLOCK_THREADS*(ITEMS_PER_THREAD + 1)
    };

    using BlockRadixSortT = hipcub::BlockRadixSort<uint32_t, BLOCK_THREADS, ITEMS_PER_THREAD, uint32_t>;
    using key_array_t = uint32_t[ITEMS_PER_THREAD];
    using value_array_t = uint32_t[ITEMS_PER_THREAD];

    l_extendedprice_buffer += blockIdx.x*BUFFER_SIZE;
    l_discount_buffer += blockIdx.x*BUFFER_SIZE;

    __shared__ uint32_t l_partkey_buffer[BUFFER_SIZE];
    __shared__ uint32_t lineitem_buffer_pos[BUFFER_SIZE];
    __shared__ int buffer_idx;

    __shared__ uint32_t fully_occupied_warps;
    __shared__ uint32_t exhausted_warps;

    __shared__ union {
        typename BlockRadixSortT::TempStorage temp_storage;
    } temp_union;

uint32_t max_partkey = 0;

    const int lane_id = threadIdx.x % 32;
    const int warp_id = threadIdx.x / 32;
    const int max_reuse = warp_id*ITEMS_PER_WARP;
    const uint32_t right_mask = __funnelshift_l(FULL_MASK, 0, lane_id);

    const unsigned tile_size = min(lineitem_size, (lineitem_size + gridDim.x - 1) / gridDim.x);
    unsigned tid = blockIdx.x*tile_size; // first tid where the first thread of a block starts scanning
    const unsigned tid_limit = min(tid + tile_size, lineitem_size); // marks the end of each tile
    tid += threadIdx.x; // each thread starts at it's correponding offset

//if (lane_id == 0 && warp_id == 0) printf("lineitem_size: %u, gridDim.x: %u, tile_size: %u\n", lineitem_size, gridDim.x, tile_size);

    uint32_t l_shipdate;
    uint32_t l_partkey;
    int64_t l_extendedprice;
    int64_t l_discount;

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    // initialize shared variables
    if (warp_id == 0 && lane_id == 0) {
        buffer_idx = 0;
        fully_occupied_warps = 0;
        exhausted_warps = 0;
    }
    __syncthreads(); // ensure that all shared variables are initialized

    uint32_t unexhausted_lanes = FULL_MASK; // lanes which can still fetch new tuples

    while (exhausted_warps < WARPS_PER_BLOCK || buffer_idx > 0) {
        // number of items stored in the buffer by this warp
        int warp_items = min(ITEMS_PER_WARP, max(0, buffer_idx - max_reuse));
//if (lane_id == 0) printf("warp: %d reuse: %u\n", warp_id, warp_items);

#ifdef MEASURE_CYCLES
        const auto scan_t1 = clock64();
#endif
        while (unexhausted_lanes && warp_items < ITEMS_PER_WARP) {
            int active = tid < tid_limit;

            // fetch attributes
            if (active) {
                l_shipdate = lineitem->l_shipdate[tid];
            }

            // filter predicate
            active = active && l_shipdate >= lower_shipdate && l_shipdate < upper_shipdate;

            // fetch remaining attributes
            if (active) {
                l_partkey = lineitem->l_partkey[tid];
                l_extendedprice = lineitem->l_extendedprice[tid];
                l_discount = lineitem->l_discount[tid];
            }

            // negotiate buffer target positions among all threads in this warp
            const uint32_t active_mask = __ballot_sync(FULL_MASK, active);
            const auto item_cnt = __popc(active_mask);
            warp_items += item_cnt;
            uint32_t dest_idx = 0;
            if (lane_id == 0) {
                dest_idx = atomicAdd(&buffer_idx, item_cnt);
 //atomicAdd(&debug_cnt, item_cnt);
            }
            dest_idx = __shfl_sync(FULL_MASK, dest_idx, 0); // propagate the first buffer target index
            dest_idx += __popc(active_mask & right_mask); // add each participating thread's offset

            // matrialize attributes
            if (active) {
                lineitem_buffer_pos[dest_idx] = dest_idx;
                l_partkey_buffer[dest_idx] = l_partkey;
                l_discount_buffer[dest_idx] = l_discount;
                l_extendedprice_buffer[dest_idx] = l_extendedprice;
                max_partkey = (l_partkey > max_partkey) ? l_partkey : max_partkey;
            }

            unexhausted_lanes = __ballot_sync(FULL_MASK, tid < tid_limit);
            if (unexhausted_lanes == 0 && lane_id == 0) {
                atomicInc(&exhausted_warps, UINT_MAX);
            }

            tid += BLOCK_THREADS; // each tile is organized as a consecutive succession of elements belonging to the current thread block
        }

        if (lane_id == 0 && warp_items >= ITEMS_PER_WARP) {
            atomicInc(&fully_occupied_warps, UINT_MAX);
        }
#ifdef MEASURE_CYCLES
        __syncwarp();
        const auto scan_t2 = clock64();
        if (lane_id == 0) {
            atomicAdd(&scan_cycles, (unsigned long long)scan_t2 - scan_t1);
        }
#endif

#ifdef MEASURE_CYCLES
        const auto sync_t1 = clock64();
#endif
        __syncthreads(); // wait until all threads have gathered enough elements
#ifdef MEASURE_CYCLES
        __syncwarp();
        const auto sync_t2 = clock64();
        if (lane_id == 0) {
            atomicAdd(&sync_cycles, (unsigned long long)sync_t2 - sync_t1);
        }
#endif


#ifndef SKIP_SORT
#ifdef MEASURE_CYCLES
        const auto sort_t1 = clock64();
#endif
        if (fully_occupied_warps == WARPS_PER_BLOCK) {
//if (warp_id == 0 && lane_id == 0) printf("=== sorting... ===\n");

            const unsigned first_offset = max(0, static_cast<int>(buffer_idx) - ITEMS_PER_BLOCK);

//if (warp_id == 0 && lane_id == 0) printf("=== first_offset: %u\n", first_offset);
            uint32_t* thread_keys_raw = reinterpret_cast<uint32_t*>(&l_partkey_buffer[threadIdx.x*ITEMS_PER_THREAD + first_offset]);
            uint32_t* thread_values_raw = reinterpret_cast<uint32_t*>(&lineitem_buffer_pos[threadIdx.x*ITEMS_PER_THREAD + first_offset]);
            key_array_t& thread_keys = reinterpret_cast<key_array_t&>(*thread_keys_raw);
            value_array_t& thread_values = reinterpret_cast<value_array_t&>(*thread_values_raw);

            BlockRadixSortT(temp_union.temp_storage).SortDescending(thread_keys, thread_values, 4, 22);
             __syncthreads();
        }
#ifdef MEASURE_CYCLES
        __syncwarp();
        const auto sort_t2 = clock64();
        if (lane_id == 0) {
            atomicAdd(&sort_cycles, (unsigned long long)sort_t2 - sort_t1);
        }
#endif
#endif

#if 1
        // empty buffer
        for (unsigned i = 0u; i < ITEMS_PER_THREAD; ++i) {
            unsigned old;
            if (lane_id == 0) {
                // T atomic_sub_safe(T* address, T val)
                old = atomic_sub_safe(&buffer_idx, 32);
            }
            old = __shfl_sync(FULL_MASK, old, 0);
            const auto acquired_cnt = min(old, 32);
            const auto first_pos = old - acquired_cnt;
//if (lane_id == 0) printf("warp: %d iteration: %d - actual_count: %u\n", warp_id, i, actual_count);
//if (lane_id == 0) atomicAdd(&debug_cnt, acquired_cnt);

            if (acquired_cnt == 0u) break;

            bool active = lane_id < acquired_cnt;

            uint32_t assoc_pos = 0u;
            key_t l_partkey;
            if (active) {
                assoc_pos = lineitem_buffer_pos[first_pos + acquired_cnt - 1 - lane_id];
                l_partkey = l_partkey_buffer[first_pos + acquired_cnt - 1 - lane_id];
//printf("warp: %d lane: %d - tid: %u l_partkey: %u\n", warp_id, lane_id, assoc_pos, l_partkey);
            }

#ifdef MEASURE_CYCLES
            const auto lookup_t1 = clock64();
#endif
            payload_t tid_b = index_structure.cooperative_lookup(active, l_partkey);
#ifdef MEASURE_CYCLES
            __syncwarp();
            const auto lookup_t2 = clock64();
            if (lane_id == 0) {
                atomicAdd(&lookup_cycles, (unsigned long long)lookup_t2 - lookup_t1);
            }
#endif

            active = active && (tid_b != invalid_tid);

            if (active) {
                const auto summand = l_extendedprice_buffer[assoc_pos] * (100 - l_discount_buffer[assoc_pos]);
                sum2 += summand;

                const char* type = reinterpret_cast<const char*>(&part->p_type[tid_b]); // FIXME relies on undefined behavior
                if (my_strcmp(type, "PROMO", 5) == 0) {
                    sum1 += summand;
                }
            }

//printf("warp: %d lane: $d - element: %u\n", warp_id, lane_id, );

        }
#else
        // discard elements
        if (lane_id == 0) buffer_idx = 0;
#endif

        // prepare next iteration
        if (lane_id == 0) {
            fully_occupied_warps = 0;
        }

        __syncthreads();
    }

    // finalize
    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}







__global__ void ij_join_kernel(const lineitem_table_plain_t* __restrict__ lineitem, const part_table_plain_t* __restrict__ part, const JoinEntry* __restrict__ join_entries, size_t n) {
    int64_t sum1 = 0;
    int64_t sum2 = 0;
    const char* prefix = "PROMO";

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        const auto lineitem_tid = join_entries[i].lineitem_tid;
        const auto part_tid = join_entries[i].part_tid;

        const auto extendedprice = lineitem->l_extendedprice[lineitem_tid];
        const auto discount = lineitem->l_discount[lineitem_tid];
        const auto summand = extendedprice * (100 - discount);
        sum2 += summand;

        const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
//        printf("type: %s\n", type);
        if (my_strcmp(type, prefix, 5) == 0) {
            sum1 += summand;
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    //__reduce_add_sync() requires compute capability 8
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}

template<class IndexType>
struct helper {
    IndexType index_structure;

    unsigned lineitem_size;
    lineitem_table_plain_t* lineitem_device;
    std::unique_ptr<lineitem_table_plain_t> lineitem_device_ptrs;

    unsigned part_size;
    part_table_plain_t* part_device;
    std::unique_ptr<part_table_plain_t> part_device_ptrs;

    void load_database(const std::string& path) {
        Database db;
        load_tables(db, path);
        if (sort_indexed_relation) {
            printf("sorting part relation...\n");
            sort_relation(db.part);
        }
        lineitem_size = db.lineitem.l_orderkey.size();
        part_size = db.part.p_partkey.size();

        {
            using namespace std;
            const auto start = chrono::high_resolution_clock::now();
            device_table_allocator<int> a;
            //std::tie(lineitem_device, lineitem_device_ptrs) = copy_relation<vector_copy_policy>(db.lineitem);
            std::tie(lineitem_device, lineitem_device_ptrs) = migrate_relation(db.lineitem, a);
            //std::tie(part_device, part_device_ptrs) = copy_relation<vector_copy_policy>(db.part);
            std::tie(part_device, part_device_ptrs) = migrate_relation(db.part, a);
            const auto finish = chrono::high_resolution_clock::now();
            const auto d = chrono::duration_cast<chrono::milliseconds>(finish - start).count();
            std::cout << "transfer time: " << d << " ms\n";
        }

#ifndef USE_HJ
        index_structure.construct(db.part.p_partkey, part_device_ptrs->p_partkey);
#endif
    }

#ifdef USE_HJ
    void run_hj() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        LinearProbingHashTable<uint32_t, size_t> ht(part_size);
        int num_blocks = (part_size + block_size - 1) / block_size;
        hj_build_kernel<<<num_blocks, block_size>>>(part_size, part_device, ht.deviceHandle);

        //num_blocks = 32*num_sms;
        num_blocks = (lineitem_size + block_size - 1) / block_size;
        hj_probe_kernel<<<num_blocks, block_size>>>(lineitem_size, part_device, lineitem_device, ht.deviceHandle);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }
#endif

    void run_ij() {
        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_full_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, part_device, index_structure.device_index);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

    void run_ij_buffer() {
        using namespace std;

        decltype(output_index) matches1 = 0;

        enum { BLOCK_THREADS = 256, ITEMS_PER_THREAD = 10 }; // TODO optimize

        JoinEntry* join_entries1;
        hipMalloc(&join_entries1, sizeof(JoinEntry)*lineitem_size);

        int num_sms;
        hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
        int num_blocks = num_sms*4; // TODO


        int buffer_size = num_blocks*BLOCK_THREADS*(ITEMS_PER_THREAD + 1);
        int64_t* l_extendedprice_buffer;
        int64_t* l_discount_buffer;
        hipMalloc(&l_extendedprice_buffer, sizeof(decltype(*l_extendedprice_buffer))*buffer_size);
        hipMalloc(&l_discount_buffer, sizeof(decltype(*l_discount_buffer))*buffer_size);

        const auto kernelStart = std::chrono::high_resolution_clock::now();

        ij_full_kernel_2<BLOCK_THREADS, ITEMS_PER_THREAD><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, part_device, part_size, index_structure.device_index, l_extendedprice_buffer, l_discount_buffer);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

    void run_two_phase_ij_plain() {
        JoinEntry* join_entries;
        hipMalloc(&join_entries, sizeof(JoinEntry)*lineitem_size);

        const auto kernelStart = std::chrono::high_resolution_clock::now();

        int num_blocks = (part_size + block_size - 1) / block_size;
        ij_lookup_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries);
        hipDeviceSynchronize();

        decltype(output_index) matches;
        hipError_t error = hipMemcpyFromSymbol(&matches, HIP_SYMBOL(output_index), sizeof(matches), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        //printf("join matches: %u\n", matches);

        num_blocks = (lineitem_size + block_size - 1) / block_size;
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries, matches);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

    void compare_join_results(JoinEntry* ref, unsigned ref_size, JoinEntry* actual, unsigned actual_size) {
        std::unordered_map<uint32_t, uint32_t> map;
        for (unsigned i = 0; i < ref_size; ++i) {
            if (map.count(ref[i].lineitem_tid) > 0) {
                std::cerr << "lineitem tid " << ref[i].lineitem_tid << " already in map" << std::endl;
                exit(0);
            }
            map.emplace(ref[i].lineitem_tid, ref[i].part_tid);
        }
        for (unsigned i = 0; i < actual_size; ++i) {
            auto it = map.find(actual[i].lineitem_tid);
            if (it != map.end()) {
                if (it->second != actual[i].part_tid) {
                    std::cerr << "part tid " << actual[i].part_tid << " expected " << it->second << std::endl;
                }
            } else {
                std::cerr << "lineitem tid " << actual[i].lineitem_tid << " not in reference" << std::endl;
            }
        }
    }

    void run_two_phase_ij_buffer_debug() {
        decltype(output_index) matches1 = 0;
        decltype(output_index) matches2 = 0;
        decltype(output_index) zero = 0;

        enum { BLOCK_THREADS = 128, ITEMS_PER_THREAD = 8 }; // TODO optimize

        JoinEntry* join_entries1;
        hipMallocManaged(&join_entries1, sizeof(JoinEntry)*lineitem_size);

        int num_sms;
        hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
        int num_blocks = num_sms*2; // TODO

        const auto start1 = std::chrono::high_resolution_clock::now();
        ij_lookup_kernel_3<BLOCK_THREADS, ITEMS_PER_THREAD><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries1);
        hipDeviceSynchronize();

        hipError_t error = hipMemcpyFromSymbol(&matches1, HIP_SYMBOL(output_index), sizeof(matches1), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        printf("join matches1: %u\n", matches1);
        printf("debug_cnt: %u\n", debug_cnt);

        error = hipMemcpyToSymbol(HIP_SYMBOL(output_index), &zero, sizeof(zero), 0, hipMemcpyHostToDevice);
        assert(error == hipSuccess);
        JoinEntry* join_entries2;
        hipMallocManaged(&join_entries2, sizeof(JoinEntry)*lineitem_size);
        num_blocks = (part_size + block_size - 1) / block_size;
        ij_lookup_kernel<<<num_blocks, block_size>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries2);
        hipDeviceSynchronize();

        error = hipMemcpyFromSymbol(&matches2, HIP_SYMBOL(output_index), sizeof(matches2), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        printf("join matches2: %u\n", matches2);

        compare_join_results(join_entries2, matches2, join_entries1, matches1);
        compare_join_results(join_entries1, matches1, join_entries2, matches2);

        const auto d1 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start1).count()/1000.;
        std::cout << "kernel time: " << d1 << " ms\n";

        num_blocks = (lineitem_size + block_size - 1) / block_size;

        const auto start2 = std::chrono::high_resolution_clock::now();
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries1, matches1);
        hipDeviceSynchronize();

        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = std::chrono::duration_cast<std::chrono::microseconds>(kernelStop - start2).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
    }

    void run_two_phase_ij_buffer() {
        using namespace std;

        decltype(output_index) matches1 = 0;

        enum { BLOCK_THREADS = 256, ITEMS_PER_THREAD = 10 }; // TODO optimize

        JoinEntry* join_entries1;
        hipMalloc(&join_entries1, sizeof(JoinEntry)*lineitem_size);

        int num_sms;
        hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
        int num_blocks = num_sms*4; // TODO

        const auto start1 = std::chrono::high_resolution_clock::now();
        //ij_lookup_kernel<<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries1);
        //ij_lookup_kernel_2<BLOCK_THREADS, ITEMS_PER_THREAD><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries1);
        ij_lookup_kernel_3<BLOCK_THREADS, ITEMS_PER_THREAD><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries1);
        //ij_lookup_kernel_4<BLOCK_THREADS><<<num_blocks, BLOCK_THREADS>>>(lineitem_device, lineitem_size, index_structure.device_index, join_entries1);
        hipDeviceSynchronize();
        const auto d1 = chrono::duration_cast<chrono::microseconds>(std::chrono::high_resolution_clock::now() - start1).count()/1000.;
        std::cout << "kernel time: " << d1 << " ms\n";

        hipError_t error = hipMemcpyFromSymbol(&matches1, HIP_SYMBOL(output_index), sizeof(matches1), 0, hipMemcpyDeviceToHost);
        assert(error == hipSuccess);
        printf("join matches1: %u\n", matches1);

        num_blocks = (lineitem_size + block_size - 1) / block_size;

        const auto start2 = std::chrono::high_resolution_clock::now();
        ij_join_kernel<<<num_blocks, block_size>>>(lineitem_device, part_device, join_entries1, matches1);
        hipDeviceSynchronize();
        const auto kernelStop = std::chrono::high_resolution_clock::now();
        const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - start2).count()/1000.;
        std::cout << "kernel time: " << kernelTime << " ms\n";
        std::cout << "complete time: " << d1 + kernelTime << " ms\n";
    }
};

template<class IndexType>
void load_and_run_ij(const std::string& path, bool as_full_pipline_breaker) {
    if (prefetch_index) { throw "not implemented"; }

    helper<IndexType> h;
    h.load_database(path);
    if (as_full_pipline_breaker) {
        printf("full pipline breaker\n");
        h.run_two_phase_ij_buffer();
    } else {
        //h.run_ij();
        h.run_ij_buffer();
    }
}

int main(int argc, char** argv) {
    using namespace std;

    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);// devId);

#ifdef USE_HJ
    if (argc != 2) {
        printf("%s <tpch dataset path>\n", argv[0]);
        return 0;
    }

    helper<int> h;
    h.load_database(argv[1]);
    //std::getc(stdin);
    h.run_hj();
#else
    if (argc < 3) {
        printf("%s <tpch dataset path> <index type: {0: btree, 1: harmonia, 2: radixspline, 3: lowerbound> <1: full pipline breaker>\n", argv[0]);
        return 0;
    }
    enum IndexType : unsigned { btree, harmonia, radixspline, lowerbound, nop } index_type { static_cast<IndexType>(std::stoi(argv[2])) };
    bool full_pipline_breaker = (argc < 4) ? false : std::stoi(argv[3]) != 0;

#ifdef SKIP_SORT
    std::cout << "skip sort step: yes" << std::endl;
#else
    std::cout << "skip sort step: no" << std::endl;
#endif

    switch (index_type) {
        case IndexType::btree: {
            printf("using btree\n");
            using index_type = btree_index<indexed_t, payload_t, device_index_allocator, host_allocator>;
            load_and_run_ij<index_type>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::harmonia: {
            printf("using harmonia\n");
            using index_type = harmonia_index<indexed_t, payload_t, device_index_allocator, host_allocator>;
            load_and_run_ij<index_type>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::radixspline: {
            printf("using radixspline\n");
            using index_type = radix_spline_index<indexed_t, payload_t, device_index_allocator, host_allocator>;
            load_and_run_ij<index_type>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::lowerbound: {
            printf("using lower bound search\n");
            using index_type = lower_bound_index<indexed_t, payload_t, device_index_allocator, host_allocator>;
            load_and_run_ij<index_type>(argv[1], full_pipline_breaker);
            break;
        }
        case IndexType::nop: {
            printf("using no_op_index\n");
            using index_type = no_op_index<indexed_t, payload_t, device_index_allocator, host_allocator>;
            load_and_run_ij<index_type>(argv[1], full_pipline_breaker);
            break;
        }
        default:
            std::cerr << "unknown index type: " << index_type << std::endl;
            return 0;
    }
#endif

/*
    printf("sum1: %lu\n", globalSum1);
    printf("sum2: %lu\n", globalSum2);
*/
    const int64_t result = 100*(globalSum1*1'000)/(globalSum2/1'000);
    printf("%ld.%ld\n", result/1'000'000, result%1'000'000);

    std::cout << std::setprecision(2) << std::scientific
        << "scan_cycles: " << (double)scan_cycles
        << "; sync_cycles: " << (double)sync_cycles
        << "; sort_cycles: " << (double)sort_cycles
        << "; lookup_cycles: " << (double)lookup_cycles 
        << std::endl; 

    hipDeviceReset();

    return 0;
}
