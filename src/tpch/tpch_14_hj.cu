#include "hip/hip_runtime.h"
#include "tpch_14_common.cuh"

#include "common.hpp"
#include "cuda_utils.cuh"
#include "LinearProbingHashTable.cuh"

__device__ unsigned int count = 0;
__managed__ int tupleCount;

using device_ht_t = LinearProbingHashTable<uint32_t, size_t>::DeviceHandle;

__global__ void hj_build_kernel(size_t n, const part_table_plain_t* part, device_ht_t ht) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (size_t i = index; i < n; i += stride) {
        ht.insert(part->p_partkey[i], i);
    }
}

__managed__ int64_t globalSum1 = 0;
__managed__ int64_t globalSum2 = 0;

__global__ void hj_probe_kernel(size_t n, const part_table_plain_t* __restrict__ part, const lineitem_table_plain_t* __restrict__ lineitem, device_ht_t ht) {
    const char* prefix = "PROMO";

    int64_t sum1 = 0;
    int64_t sum2 = 0;

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (lineitem->l_shipdate[i] < lower_shipdate ||
            lineitem->l_shipdate[i] >= upper_shipdate) {
            continue;
        }

        size_t part_tid;
        bool match = ht.lookup(lineitem->l_partkey[i], part_tid);
        // TODO use lane refill
        if (match) {
            const auto extendedprice = lineitem->l_extendedprice[i];
            const auto discount = lineitem->l_discount[i];
            const auto summand = extendedprice * (100 - discount);
            sum2 += summand;

            const char* type = reinterpret_cast<const char*>(&part->p_type[part_tid]); // FIXME relies on undefined behavior
            if (device_strcmp(type, prefix, 5) == 0) {
                sum1 += summand;
            }
        }
    }

    // reduce both sums
    #pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum1 += __shfl_down_sync(FULL_MASK, sum1, offset);
        sum2 += __shfl_down_sync(FULL_MASK, sum2, offset);
    }
    if (lane_id() == 0) {
        atomicAdd((unsigned long long int*)&globalSum1, (unsigned long long int)sum1);
        atomicAdd((unsigned long long int*)&globalSum2, (unsigned long long int)sum2);
    }
}
