#include "hip/hip_runtime.h"
//#include <numa-gpu/sql-ops/include/gpu_radix_partition.h>
#include "gpu_radix_partition.cuh"

// Combine the following to file into this single compilation unite in order to avoid using Relocatable Device Code (-rdc=true).
#include <numa-gpu/sql-ops/cudautils/gpu_common.cu>
#include <numa-gpu/sql-ops/cudautils/radix_partition.cu>

__global__ void partitioned_consumer_assign_tasks(partitioned_consumer_assign_tasks_args args) {
    const auto fanout = 1U << args.radix_bits;

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        const uint32_t rel_size = args.rel_length - args.rel_padding_length*fanout;
        const uint32_t avg_task_size = (rel_size + gridDim.x - 1U) / gridDim.x;

        args.task_assignment[0] = 0U;
        uint32_t task_id = 1U;
        uint32_t task_size = 0U;
        for (uint32_t p = 0U; p < fanout && task_id < gridDim.x; ++p) {
            const uint32_t partition_upper = (p + 1U < fanout) ? args.rel_partition_offsets[p + 1U] - args.rel_padding_length : args.rel_length;
            const uint32_t partition_size = static_cast<uint32_t>(partition_upper - args.rel_partition_offsets[p]);

            task_size += partition_size;
            if (task_size >= avg_task_size) {
                args.task_assignment[task_id] = p + 1U;
#ifdef DEBUG
        printf("Assigning partitions [%u, %u] to block %d\n",
               args.task_assignments[task_id - 1],
               args.task_assignments[task_id], task_id);
#endif

                task_size = 0U;
                task_id += 1;
            }
        }

        for (uint32_t i = task_id; i <= gridDim.x; ++i) {
            args.task_assignment[i] = fanout;
        }
    }
}
