#include "btree.cuh"

#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <numeric>
#include <sys/types.h>
#include <chrono>

#include "zipf.hpp"

using namespace std;

static constexpr unsigned numElements = 1e7;

int main() {

    std::vector<btree::key_t> keys(numElements);
    std::iota(keys.begin(), keys.end(), 0);

//    auto tree = btree::construct_dense(1e6, 0.7);
    auto tree = btree::construct(keys, 0.7);
    for (unsigned i = 0; i < numElements; ++i) {
        //printf("lookup %d\n", i);
        btree::payload_t value;
        bool found = btree::lookup(tree, keys[i], value);
        if (!found) throw 0;
    }

    int blockSize = 32;
    int numBlocks = (numElements + blockSize - 1) / blockSize;
/*
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int numBlocks = 32*numSMs;*/
    printf("numblocks: %d\n", numBlocks);

    // shuffle keys
    auto rng = std::default_random_engine {};
    std::shuffle(std::begin(keys), std::end(keys), rng);
    // TODO zipfian lookup patterns

    btree::key_t* lookupKeys;
    hipMalloc(&lookupKeys, numElements*sizeof(key_t));
    // TODO shuffle keys/Zipfian lookup patterns
    hipMemcpy(lookupKeys, keys.data(), numElements*sizeof(key_t), hipMemcpyHostToDevice);
    btree::payload_t* tids;
    hipMallocManaged(&tids, numElements*sizeof(decltype(tids)));

//    btree::prefetchTree(tree);

    auto start = std::chrono::high_resolution_clock::now();
    btree::cuda::btree_bulk_lookup<<<numBlocks, blockSize>>>(tree, numElements, lookupKeys, tids);
    hipDeviceSynchronize();
    auto kernelStop = std::chrono::high_resolution_clock::now();
    auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - start).count()/1000.;
    std::cout << "Kernel time: " << kernelTime << " ms\n";
    std::cout << "GPU MOps: " << (numElements/1e6)/(kernelTime/1e3) << endl;

/*
    for (unsigned i = 0; i < numElements; ++i) {
        printf("tid: %lu\n", reinterpret_cast<uint64_t>(tids[i]));
    }
*/

    start = std::chrono::high_resolution_clock::now();
    for (unsigned i = 0; i < numElements; ++i) {
        btree::payload_t value;
        bool found = btree::lookup(tree, keys[i], value);
        if (!found) throw 0;
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto cpuTime = chrono::duration_cast<chrono::microseconds>(stop - start).count()/1000.;
    std::cout << "CPU time: " << cpuTime << " ms\n";
    std::cout << "CPU MOps: " << (numElements/1e6)/(cpuTime/1e3) << endl;

    return 0;
}
