#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>
#include <numeric>
#include <chrono>
#include <cstring>
#include <stdexcept>

#include "rs/multi_map.h"

namespace rs {

using rs_rt_entry_t = uint32_t;

template<class Key>
struct RawRadixSpline {
    using key_t = Key;
    using spline_point_t = rs::Coord<key_t>;

    key_t min_key_;
    key_t max_key_;
    size_t num_keys_;
    size_t num_radix_bits_;
    size_t num_shift_bits_;
    size_t max_error_;

    std::vector<rs_rt_entry_t> radix_table_;
    std::vector<spline_point_t> spline_points_;
};

template<class Key>
struct DeviceRadixSpline {
    using key_t = Key;
    using spline_point_t = rs::Coord<key_t>;

    key_t min_key_;
    key_t max_key_;
    size_t num_keys_;
    size_t num_radix_bits_;
    size_t num_shift_bits_;
    size_t max_error_;

    rs_rt_entry_t* radix_table_;
    spline_point_t* spline_points_;
};

template<class Key>
auto build_radix_spline(const std::vector<Key>& keys) {
    auto min = keys.front();
    auto max = keys.back();
    rs::Builder<Key> rsb(min, max);
    for (const auto& key : keys) rsb.AddKey(key);
    rs::RadixSpline<Key> rs = rsb.Finalize();
    return rs;
}

template<class Policy, class Key>
DeviceRadixSpline<Key>* copy_radix_spline(const rs::RadixSpline<Key>& radixSpline) {
    static Policy f;

    static DeviceRadixSpline<Key> tmp;
    const RawRadixSpline<Key>* rrs = reinterpret_cast<const RawRadixSpline<Key>*>(&radixSpline);
    std::memcpy(&tmp, &radixSpline, sizeof(DeviceRadixSpline<Key>));

    // copy radix table
    tmp.radix_table_ = f(rrs->radix_table_);

    // copy spline points
    tmp.spline_points_ = f(rrs->spline_points_);

    DeviceRadixSpline<Key>* d_rs;
    hipMalloc(&d_rs, sizeof(DeviceRadixSpline<Key>));
    hipMemcpy(d_rs, &tmp, sizeof(DeviceRadixSpline<Key>), hipMemcpyHostToDevice);
    return d_rs;
}

namespace cuda {

template<typename T, typename P>
__device__ unsigned lower_bound(const T& key, const T* arr, const unsigned size) {
    unsigned lower = 0;
    unsigned upper = size;
    do {
        unsigned mid = ((upper - lower) / 2) + lower;
        int c = cmp(arr[mid], key); // a < b
        if (key < arr[mid]) {
            upper = mid;
        } else if (key > arr[mid]) {
            lower = mid + 1;
        } else {
            return mid;
        }
    } while (lower < upper);
    return lower;
}

template<typename T1, typename T2, typename P>
__device__ unsigned lower_bound(const T1& key, const T2* arr, const unsigned size, P cmp) {
    unsigned lower = 0;
    unsigned count = size;
    while (count > 0) {
        unsigned step = count / 2;
        unsigned mid = lower + step;
        if (cmp(arr[mid], key)) {
            lower = mid + 1;
            count -= step + 1;
        } else {
            count = step;
        }
    }
    return lower;
}

__global__ void do_lower_bound(const int* arr, const unsigned size) {
    lower_bound(0, arr, size, [] (const auto& a, const auto& b) {
        return a < b;
    });
}

template<class Key>
__device__ unsigned get_spline_segment(const DeviceRadixSpline<Key>* rs, const Key key) {
    const auto prefix = (key - rs->min_key_) >> rs->num_shift_bits_;

    const uint32_t begin = rs->radix_table_[prefix];
    const uint32_t end = rs->radix_table_[prefix + 1];

    // TODO measure linear search for narrow ranges as in the reference implementation

    const auto range_size = end - begin;
    const auto lb = begin + lower_bound(key, rs->spline_points_ + begin, range_size, [] (const auto& coord, const Key key) {
        return coord.x < key;
    });
//    printf("key: %lu, lb: %u\n", key, lb);
    return lb;
}

template<class Key>
__device__ double get_estimate(const DeviceRadixSpline<Key>* rs, const Key key) {
    if (key <= rs->min_key_) return 0;
    if (key >= rs->max_key_) return rs->num_keys_ - 1;

    // find spline segment
    const unsigned index = get_spline_segment(rs, key);
    const auto& down = rs->spline_points_[index - 1];
    const auto& up = rs->spline_points_[index];

    // slope
    const double x_diff = up.x - down.x;
    const double y_diff = up.y - down.y;
    const double slope = y_diff / x_diff;

    // interpolate
    const double key_diff = key - down.x;
    return key_diff * slope + down.y;
}

}

}
