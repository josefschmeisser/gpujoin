#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <numeric>
#include <chrono>
#include <memory>

#include "zipf.hpp"

#include "btree.cuh"
#include "btree.cu"

using namespace std;

static constexpr unsigned maxRepetitions = 10;
static constexpr unsigned numElements = 1e8;

using namespace btree;
using namespace btree::cuda;

__global__ void btree_bulk_lookup(Node* tree, unsigned n, btree::key_t* keys, payload_t* tids) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        //tids[i] = btree_lookup(tree, keys[i]);
        tids[i] = btree::cuda::btree_lookup_with_hints(tree, keys[i]);
    }
}

int main() {

    std::vector<btree::key_t> keys(numElements);
    std::iota(keys.begin(), keys.end(), 0);

//    auto tree = btree::construct_dense(1e6, 0.7);
    auto tree = btree::construct(keys, 0.7);
    for (unsigned i = 0; i < numElements; ++i) {
        //printf("lookup %d\n", i);
        btree::payload_t value;
        bool found = btree::lookup(tree, keys[i], value);
        if (!found) throw 0;
    }

    int blockSize = 32;
    int numBlocks = (numElements + blockSize - 1) / blockSize;
/*
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int numBlocks = 32*numSMs;*/
    printf("numblocks: %d\n", numBlocks);

    // shuffle keys
    auto rng = std::default_random_engine {};
    std::shuffle(std::begin(keys), std::end(keys), rng);
    // TODO zipfian lookup patterns

    btree::key_t* lookupKeys;
    hipMalloc(&lookupKeys, numElements*sizeof(btree::key_t));
    // TODO shuffle keys/Zipfian lookup patterns
    hipMemcpy(lookupKeys, keys.data(), numElements*sizeof(btree::key_t), hipMemcpyHostToDevice);
    btree::payload_t* d_tids;
    hipMalloc(&d_tids, numElements*sizeof(decltype(d_tids)));

    btree::prefetchTree(tree, 0);

    printf("executing kernel...\n");
    const auto kernelStart = std::chrono::high_resolution_clock::now();
    for (unsigned rep = 0; rep < maxRepetitions; ++rep) {
        btree_bulk_lookup<<<numBlocks, blockSize>>>(tree, numElements, lookupKeys, d_tids);
        hipDeviceSynchronize();
    }
    const auto kernelStop = std::chrono::high_resolution_clock::now();
    const auto kernelTime = chrono::duration_cast<chrono::microseconds>(kernelStop - kernelStart).count()/1000.;
    std::cout << "Kernel time: " << kernelTime << " ms\n";
    std::cout << "GPU MOps: " << (maxRepetitions*numElements/1e6)/(kernelTime/1e3) << endl;

    // validate results
    printf("validating results...\n");
    std::unique_ptr<btree::payload_t[]> h_tids(new btree::payload_t[numElements]);
    //btree::payload_t* h_tids =
    hipMemcpy(h_tids.get(), d_tids, numElements*sizeof(decltype(d_tids)), hipMemcpyDeviceToHost);
    for (unsigned i = 0; i < numElements; ++i) {
        //printf("tid: %lu key[i]: %lu\n", reinterpret_cast<uint64_t>(h_tids[i]), keys[i]);
        if (reinterpret_cast<uint64_t>(h_tids[i]) != keys[i]) {
            printf("i: %u tid: %lu key[i]: %u\n", i, reinterpret_cast<uint64_t>(h_tids[i]), keys[i]);
            throw;
        }
    }

    return 0;

    const auto cpuStart = std::chrono::high_resolution_clock::now();
    for (unsigned i = 0; i < numElements; ++i) {
        btree::payload_t value;
        bool found = btree::lookup(tree, keys[i], value);
        if (!found) throw 0;
    }
    const auto cpuStop = std::chrono::high_resolution_clock::now();
    const auto cpuTime = chrono::duration_cast<chrono::microseconds>(cpuStop - cpuStart).count()/1000.;
    std::cout << "CPU time: " << cpuTime << " ms\n";
    std::cout << "CPU MOps: " << (numElements/1e6)/(cpuTime/1e3) << endl;

    return 0;
}
