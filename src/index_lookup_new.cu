#include "hip/hip_runtime.h"
#include "index_lookup.cuh"

#include <cmath>
#include <cstdio>
#include <map>
#include <stdexcept>
#include <string>
#include <memory>

#include <oneapi/tbb/parallel_sort.h>

#include "index_lookup_config.hpp"
#include "index_lookup_common.cuh"
#include "index_lookup_partitioning.cuh"
#include "device_properties.hpp"
#include "indexes.hpp"
#include "measuring.hpp"

#ifdef NRDC
#include "index_lookup_partitioning.cu"
#endif

using namespace measuring;


query_data::query_data() {
    auto& config = get_experiment_config();

    // generate datasets
    printf("generating datasets...\n");
    indexed.resize(config.num_elements);
    lookup_keys.resize(config.num_lookups);
    generate_datasets<index_key_t>(config.dataset, config.max_bits, indexed, config.lookup_pattern, config.zipf_factor, lookup_keys);
    if (config.sorted_lookups) {
        printf("sorting lookups...\n");
        oneapi::tbb::parallel_sort(lookup_keys.begin(), lookup_keys.end());
    }
    //std::cout << "lookups: " << stringify(lookup_keys.begin(), lookup_keys.end()) << std::endl;

    // indexed is guaranteed to be sorted
    dataset_max_bits = static_cast<unsigned>(std::log2(indexed.back()));
    if (config.partitioning_approach_dynamic_bit_range) {
        config.partitioning_approach_ignore_bits = (dataset_max_bits > radix_bits) ? dataset_max_bits - radix_bits : config.partitioning_approach_ignore_bits;
        printf("config.partitioning_approach_ignore_bits: %d\n", config.partitioning_approach_ignore_bits);
    }

    // allocate result vector
    d_tids = create_device_array<value_t>(config.num_lookups);

    // create gpu accessible vectors
    indexed_allocator_t indexed_allocator;
    d_indexed = create_device_array_from(indexed, indexed_allocator);
    lookup_keys_allocator_t lookup_keys_allocator;
    d_lookup_keys = create_device_array_from(lookup_keys, lookup_keys_allocator);

    // finalize state
    printf("generating index...\n");
    create_index();
}

void query_data::create_index() {
    const auto& config = get_experiment_config();

    // allocate index structure
    switch (parse_index_type(config.index_type)) {
        case index_type_enum::btree:
            index_structure = build_index<index_key_t, btree_type>(indexed, d_indexed.data());
            break;
        case index_type_enum::harmonia:
            index_structure = build_index<index_key_t, harmonia_type>(indexed, d_indexed.data());
            break;
        case index_type_enum::binary_search:
            index_structure = build_index<index_key_t, binary_search_type>(indexed, d_indexed.data());
            break;
        case index_type_enum::radix_spline:
            index_structure = build_index<index_key_t, radix_spline_type>(indexed, d_indexed.data());
            break;
        case index_type_enum::no_op:
            index_structure = build_index<index_key_t, no_op_type>(indexed, d_indexed.data());
            break;
        default:
            assert(false);
    }
}

bool query_data::validate_results() {
    auto h_tids = d_tids.to_host_accessible();
    auto h_tids_raw = h_tids.data();

    // validate results
    printf("validating results...\n");
    for (unsigned i = 0; i < lookup_keys.size(); ++i) {
        if (h_tids_raw[i] > indexed.size()) {
            std::cerr << "invalid tid: " << h_tids_raw[i] << ", at " << i << " from " << lookup_keys.size() << std::endl;
            return false;
        }
        if (lookup_keys[i] != indexed[h_tids_raw[i]]) {
            std::cerr << "lookup_keys[" << i << "]: " << lookup_keys[i] << "indexed[h_tids[" << i << "]]: " << indexed[h_tids_raw[i]] << std::endl;
            return false;
        }
    }
    printf("validation complete\n");

    return true;
}


struct abstract_approach_dispatcher {
    virtual void run(query_data& d, index_type_enum index_type) const = 0;
};

template<template<class T> class Func>
struct approach_dispatcher : public abstract_approach_dispatcher {
    void run(query_data& d, index_type_enum index_type) const override {
        switch (index_type) {
            case index_type_enum::btree:
                Func<btree_type>()(d);
                break;
            case index_type_enum::harmonia:
                Func<harmonia_type>()(d);
                break;
            case index_type_enum::binary_search:
                Func<binary_search_type>()(d);
                break;
            case index_type_enum::radix_spline:
                Func<radix_spline_type>()(d);
                break;
            case index_type_enum::no_op:
                Func<no_op_type>()(d);
                break;
            default:
                assert(false);
        }
    }
};

template<class IndexType>
struct my_approach {
    void operator()(query_data& d) {
        printf("my approach %s\n", type_name<IndexType>::value());
    }
};

template<class IndexType>
struct plain_approach {
    void operator()(query_data& d) {
        const auto& config = get_experiment_config();

        const int num_blocks = (config.num_lookups + config.block_size - 1) / config.block_size;
        printf("numblocks: %d\n", num_blocks);

        printf("executing kernel...\n");
        IndexType& index_structure = *static_cast<IndexType*>(d.index_structure.get());
        lookup_kernel<<<num_blocks, config.block_size>>>(index_structure.device_index, d.lookup_keys.size(), d.d_lookup_keys.data(), d.d_tids.data());

        hipDeviceSynchronize();
    }
};

template<class IndexType>
struct blockwise_sorting_approach {
    void operator()(query_data& d) {
        const auto& config = get_experiment_config();

        const int num_blocks = 3 * get_device_properties(0).multiProcessorCount;; // TODO optimize
        printf("numblocks: %d\n", num_blocks);

        printf("executing kernel...\n");
        IndexType& index_structure = *static_cast<IndexType*>(d.index_structure.get());
        if (config.block_size != 256) {
            std::cerr << "invalid block size for this approach" << std::endl;
            throw 0;
        }
        lookup_kernel_with_sorting_v1<256, 4><<<num_blocks, 256>>>(index_structure.device_index, d.lookup_keys.size(), d.d_lookup_keys.data(), d.d_tids.data(), d.dataset_max_bits);
        hipDeviceSynchronize();
    }
};

//static const std::map<std::string, std::unique_ptr<abstract_approach_dispatcher>> approaches {
static const std::map<std::string, std::shared_ptr<abstract_approach_dispatcher>> approaches {
    { "plain", std::make_shared<approach_dispatcher<plain_approach>>() },
    { "bws", std::make_shared<approach_dispatcher<blockwise_sorting_approach>>() },
    { "partitioning", std::make_shared<approach_dispatcher<partitioning_approach>>() }
};

static void add_index_configuration_description(std::vector<std::pair<std::string, std::string>>& pairs, const query_data& qd) {
    const auto& config = get_experiment_config();

    switch (parse_index_type(config.index_type)) {
        case index_type_enum::btree:
            pairs.emplace_back("index_lookup_algorithm", std::string(btree_type::index_configuration_t::cooperative_lookup_algorithm_type::name()));
            break;
        case index_type_enum::binary_search:
            pairs.emplace_back("index_search_algorithm", std::string(binary_search_type::index_configuration_t::cooperative_search_algorithm_type::name()));
            break;
        case index_type_enum::radix_spline:
            pairs.emplace_back("index_search_algorithm", std::string(radix_spline_type::index_configuration_t::cooperative_search_algorithm_type::name()));
            break;
    }

    pairs.emplace_back("index_size", "!int64 " + std::to_string(qd.index_structure->memory_consumption()));
}

static void create_common_experiment_description_pairs_2(std::vector<std::pair<std::string, std::string>>& pairs) {
    const auto& config = get_experiment_config();

    pairs.emplace_back(std::string("device"), std::string(get_device_properties(0).name));
    pairs.emplace_back(std::string("index_type"), config.index_type);
    pairs.emplace_back(std::string("dataset"), tmpl_to_string(config.dataset));
    pairs.emplace_back(std::string("lookup_pattern"), tmpl_to_string(config.lookup_pattern));
    pairs.emplace_back(std::string("num_elements"), "!int64 " + std::to_string(config.num_elements));
    pairs.emplace_back(std::string("num_lookups"), "!int64 " + std::to_string(config.num_lookups));
    pairs.emplace_back(std::string("sorted_lookups"), std::to_string(config.sorted_lookups));
        // allocators:
    pairs.emplace_back(std::string("host_allocator"), std::string(type_name<host_allocator_t<int>>::value()));
    pairs.emplace_back(std::string("device_index_allocator"), std::string(type_name<device_index_allocator<int>>::value()));
    pairs.emplace_back(std::string("indexed_allocator"), std::string(type_name<indexed_allocator_t>::value()));
    pairs.emplace_back(std::string("lookup_keys_allocator"), std::string(type_name<lookup_keys_allocator_t>::value()));

    if (config.dataset == dataset_type::sparse) {
        pairs.emplace_back(std::string("max_bits"), std::to_string(config.max_bits));
    }

    if (config.lookup_pattern == lookup_pattern_type::zipf) {
        pairs.emplace_back(std::string("zipf_factor"), std::to_string(config.zipf_factor));
    }

    if (config.approach == "partitioning") {
        pairs.emplace_back(std::string("partitioning_approach_ignore_bits"), std::to_string(config.partitioning_approach_ignore_bits));
        pairs.emplace_back(std::string("partitioning_approach_dynamic_bit_range"), std::to_string(config.partitioning_approach_dynamic_bit_range));
    }
}

static measuring::experiment_description create_experiment_description(const query_data& qd) {
    const auto& config = get_experiment_config();

    experiment_description r;
    r.name = "plain_lookup";
    r.approach = config.approach;

    create_common_experiment_description_pairs_2(r.other);
    add_index_configuration_description(r.other, qd);

    return r;
}

void execute_approach(std::string approach_name) {
    auto& config = get_experiment_config();

    query_data qd;

    const auto experiment_desc = create_experiment_description(qd);
    index_type_enum index_type = parse_index_type(config.index_type);
    auto validator = [&qd]() {
        return qd.validate_results();
    };
    measure(experiment_desc, [&](auto& measurement) {
        approaches.at(approach_name)->run(qd, index_type);
    }, validator);
}

void execute_benchmark_scenario(std::string scenario) {
    const auto& config = get_experiment_config();
    execute_approach(config.approach);
}

int main(int argc, char** argv) {
    parse_options(argc, argv);
    const auto& config = get_experiment_config();

    // set-up the measuring utility
    auto& measuring_config = measuring::get_settings();
    if (!config.output_file.empty()) {
        measuring_config.stdout_only = false;
        measuring_config.dest_file = config.output_file;
    } else {
        measuring_config.stdout_only = true;
    }
    measuring_config.repetitions = 10;
/*
    if (config.execute_predefined_scenario) {
        execute_benchmark_scenario();
        return;
    }*/

    execute_benchmark_scenario(config.scenario);

#if 0
    const auto experiment_desc = create_experiment_description();
    // TODO port
    std::unique_ptr<value_t[]> h_tids;
    if /*constexpr*/ (activeLanes < 32) {
        assert(false); // TODO
    } else {
        auto result = run_lookup_benchmark(experiment_desc, *index, d_lookup_keys.data(), lookup_keys.size());
        h_tids.swap(result);
    }
#endif

    return 0;
}
