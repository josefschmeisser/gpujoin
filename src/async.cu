#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <chrono>
#include <memory>

#include <cub/util_debug.cuh>

#include <numa-gpu/sql-ops/include/gpu_radix_partition.h>
#include <numa-gpu/sql-ops/cudautils/gpu_common.cu>
#include <numa-gpu/sql-ops/cudautils/radix_partition.cu>

#include "cuda_utils.cuh"
#include "cuda_allocator.hpp"
#include "numa_allocator.hpp"
#include "mmap_allocator.hpp"
#include "indexes.cuh"
#include "device_array.hpp"

#include "index_lookup_config.cuh"
#include "index_lookup_common.cuh"


static const int block_size = 128;// 64;
static const int grid_size = 10;//1;
static const uint32_t radix_bits = 6;// 10;
static const uint32_t ignore_bits = 0;//3;

template<class T> using device_allocator_t = cuda_allocator<T, cuda_allocation_type::device>;
template<class T> using device_index_allocator_t = cuda_allocator<T, cuda_allocation_type::zero_copy>;

namespace gpu_prefix_sum {

// same as in partition.rs
uint32_t fanout(uint32_t radix_bits) {
    return (1 << radix_bits);
}

template<class G, class B>
size_t state_size(G grid_size, B block_size) {
    hipDeviceProp_t device_properties;
    const auto ret = hipGetDeviceProperties(&device_properties, 0); // FIXME
    CubDebugExit(ret);

    const auto warp_size = device_properties.warpSize;
    return ((grid_size * block_size) / warp_size + warp_size);
}

} // end namespace gpu_prefix_sum


struct partition_offsets {
    device_array_wrapper<unsigned long long> offsets;
    device_array_wrapper<unsigned long long> local_offsets;

    template<class Allocator>
    partition_offsets(uint32_t max_chunks, uint32_t radix_bits, Allocator& allocator) {
        const auto chunks = 1; // we only consider contiguous histograms (at least for now)
        const auto num_partitions = gpu_prefix_sum::fanout(radix_bits);
        offsets = create_device_array<unsigned long long>(num_partitions * chunks);
        local_offsets = create_device_array<unsigned long long>(num_partitions * max_chunks);
    }
};

template<class T>
constexpr unsigned padding_length() {
    return GPU_CACHE_LINE_SIZE / sizeof(T);
}

template<class T>
struct partitioned_relation {
    device_array_wrapper<T> relation;
    device_array_wrapper<uint64_t> offsets;

    template<class Allocator>
    partitioned_relation(size_t len, uint32_t max_chunks, uint32_t radix_bits, Allocator& allocator) {
        const auto chunks = 1; // we only consider contiguous histograms (at least for now)
        const auto padding_len = ::padding_length<T>();
        const auto num_partitions = gpu_prefix_sum::fanout(radix_bits);
        const auto relation_len = len + (num_partitions * chunks) * padding_len;
printf("relation_len: %lu\n", relation_len);
        // allocate device accessible arrays
        relation = create_device_array<T>(relation_len);
        offsets = create_device_array<uint64_t>(num_partitions * chunks);
    }

    unsigned padding_length() const {
        return ::padding_length<T>();
    }
};

#if 0
const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    return 0;
}
#endif


__global__ void partition_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes)*/

}

__global__ void join_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
}

void dump_offsets(const partition_offsets& offsets) {
    auto h_offsets = offsets.offsets.to_host_accessible();
    std::cout << stringify(h_offsets.data(), h_offsets.data() + h_offsets.size()) << std::endl;
    auto h_local_offsets = offsets.local_offsets.to_host_accessible();
    std::cout << stringify(h_local_offsets.data(), h_local_offsets.data() + h_local_offsets.size()) << std::endl;
}

int main(int argc, char** argv) {
    double zipf_factor = 1.25;
    auto num_elements = default_num_elements;
    size_t num_lookups = 1000;// default_num_lookups;
    if (argc > 1) {
        std::string::size_type sz;
        num_elements = std::stod(argv[1], &sz);
    }
    std::cout << "index size: " << num_elements << std::endl;

    // generate datasets
    std::vector<index_key_t, host_allocator_t<index_key_t>> indexed, lookup_keys;
    indexed.resize(num_elements);
    lookup_keys.resize(default_num_elements);
    generate_datasets<index_key_t, index_type>(dataset_type::dense, max_bits, indexed, lookup_pattern_type::zipf, zipf_factor, lookup_keys);

    // create gpu accessible vectors
    indexed_allocator_t indexed_allocator;
    auto d_indexed = create_device_array_from(indexed, indexed_allocator);
    lookup_keys_allocator_t lookup_keys_allocator;
    auto d_lookup_keys = create_device_array_from(lookup_keys, lookup_keys_allocator);
    auto index = build_index<index_key_t, index_type>(indexed, d_indexed.data());

    // fetch device properties
    hipDeviceProp_t device_properties;
    CubDebugExit(hipGetDeviceProperties(&device_properties, 0));
    std::cout << "sharedMemPerBlock: " << device_properties.sharedMemPerBlock << std::endl;

    device_allocator_t<int> device_allocator;


/*
struct PrefixSumAndCopyWithPayloadArgs {
  // Inputs
  const void *const __restrict__ src_partition_attr;
  const void *const __restrict__ src_payload_attr;
  std::size_t const data_length;
  std::size_t const canonical_chunk_length;
  uint32_t const padding_length;
  uint32_t const radix_bits;
  uint32_t const ignore_bits;

  // State
  ScanState<unsigned long long> *const prefix_scan_state;
  unsigned long long *const __restrict__ tmp_partition_offsets;

  // Outputs
  void *const __restrict__ dst_partition_attr;
  void *const __restrict__ dst_payload_attr;
  unsigned long long *const __restrict__ partition_offsets;
};
*/

    // dummy payloads
    auto d_payloads = create_device_array<int32_t>(num_lookups);

    // allocate output arrays
    auto dst_partition_attr = create_device_array<index_key_t>(num_lookups);
    auto dst_payload_attrs = create_device_array<int32_t>(num_lookups);

    //ScanState<unsigned long long>* prefix_scan_state; // see: device_exclusive_prefix_sum_initialize
    const auto prefix_scan_state_len = gpu_prefix_sum::state_size(grid_size, block_size);
    auto prefix_scan_state = create_device_array<ScanState<unsigned long long>>(prefix_scan_state_len);

    partition_offsets offsets(grid_size, radix_bits, device_allocator);
    //partitioned_relation<index_key_t> partitioned_relation_inst(num_lookups, grid_size, radix_bits, device_allocator);
    partitioned_relation<Tuple<int32_t, int32_t>> partitioned_relation_inst(num_lookups, grid_size, radix_bits, device_allocator);
/*
    PrefixSumAndCopyWithPayloadArgs prefix_sum_and_copy_args {
        // Inputs
        d_lookup_keys.data(),
        d_payloads.data(),
        num_lookups,
        0, // not used
        partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
        // State
        prefix_scan_state.data(),
        offsets.local_offsets.data(),
        // Outputs
        dst_partition_attr.data(),
        dst_payload_attrs.data(),
        offsets.offsets.data()
    };
    */
    PrefixSumArgs prefix_sum_and_copy_args {
        // Inputs
        d_lookup_keys.data(),
        num_lookups,
        0, // not used
        partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
        // State
        prefix_scan_state.data(),
        offsets.local_offsets.data(),
        // Outputs
        offsets.offsets.data()
    };

    //__host__ ​hipError_t hipLaunchCooperativeKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream ) 
    /*
    template <typename K, typename V>
    __device__ void gpu_contiguous_prefix_sum_and_copy_with_payload(args)
    */
    hipStream_t scan_stream;
    CubDebugExit(hipStreamCreate(&scan_stream));

    const auto required_shared_mem_bytes = ((block_size + (block_size >> LOG2_NUM_BANKS)) + gpu_prefix_sum::fanout(radix_bits)) * sizeof(uint64_t);
    printf("required_shared_mem_bytes %lu\n", required_shared_mem_bytes);
    //const void* func = &gpu_contiguous_prefix_sum_and_copy_with_payload<int, int>;
    assert(required_shared_mem_bytes <= device_properties.sharedMemPerBlock);

    // prepare kernel arguments
    //auto d_prefix_sum_and_copy_args = create_device_array_from(reinterpret_cast<const uint8_t*>(&prefix_sum_and_copy_args), sizeof(prefix_sum_and_copy_args));
    void* args[1];
    args[0] = &prefix_sum_and_copy_args;// d_prefix_sum_and_copy_args.data();

    CubDebugExit(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
        //func),
//        (void*)gpu_contiguous_prefix_sum_and_copy_with_payload_int32_int32,
        //(void*)gpu_chunked_prefix_sum_int32,
        (void*)gpu_contiguous_prefix_sum_int32,
        dim3(grid_size),
        dim3(block_size),
        args,
        required_shared_mem_bytes,
        scan_stream
    ));
    hipDeviceSynchronize();
    printf("gpu_contiguous_prefix_sum_and_copy_with_payload_int32_int32 done\n");

dump_offsets(offsets);
/*
// Arguments to the partitioning function.
//
// Note that the struct's layout must be kept in sync with its counterpart in
// Rust.
struct RadixPartitionArgs {
  // Inputs
  const void *const __restrict__ join_attr_data;
  const void *const __restrict__ payload_attr_data;
  std::size_t const data_length;
  uint32_t const padding_length;
  uint32_t const radix_bits;
  uint32_t const ignore_bits;
  const unsigned long long *const __restrict__ partition_offsets;

  // State
  uint32_t *const __restrict__ tmp_partition_offsets;
  char *const __restrict__ l2_cache_buffers;
  char *const __restrict__ device_memory_buffers;
  uint64_t const device_memory_buffer_bytes;

  // Outputs
  void *const __restrict__ partitioned_relation;
};
*/

    RadixPartitionArgs radix_partition_args {
        // Inputs
        //dst_partition_attr.data(),
        //dst_payload_attrs.data(),
        d_lookup_keys.data(),
        d_payloads.data(),
        num_lookups,
        partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
//        offsets.local_offsets.data(),
        offsets.offsets.data(),
        // State
        nullptr,
        nullptr,
        nullptr,
        0,
        // Outputs
        partitioned_relation_inst.relation.data()
    };

    const auto required_shared_mem_bytes_2 = gpu_prefix_sum::fanout(radix_bits) * sizeof(uint32_t);

    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes);
    */

    //gpu_chunked_laswwc_radix_partition<<<1, 64>>>(args, );

//gpu_chunked_laswwc_radix_partition_int32_int32<<<grid_size, block_size, device_properties.sharedMemPerBlock, scan_stream>>>(radix_partition_args, device_properties.sharedMemPerBlock);

    gpu_chunked_radix_partition_int32_int32<<<grid_size, block_size, required_shared_mem_bytes_2, scan_stream>>>(radix_partition_args);
    hipDeviceSynchronize();


    printf("gpu_chunked_laswwc_radix_partition_int32_int32 done\n");

#if 0
    hipStream_t partition_stream, join_stream;
    hipStreamCreate(partition_stream);
    hipStreamCreate(join_stream);


    partition_kernel<<<1, 64, 0, partition_stream>>>();
    join_kernel<<<1, 64, 0, join_stream>>>();
    hipDeviceSynchronize();

#endif
    hipDeviceReset();

    return 0;
}
