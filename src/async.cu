#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <chrono>
#include <memory>


#include <numa-gpu/sql-ops/include/gpu_radix_partition.h>
#include <numa-gpu/sql-ops/cudautils/gpu_common.cu>
#include <numa-gpu/sql-ops/cudautils/radix_partition.cu>



#if 0
const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    return 0;
}
#endif


__global__ void partition_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes)*/

}

__global__ void join_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
}

int main(int argc, char** argv) {
    hipDeviceProp_t device_properties;
    const auto ret = hipGetDeviceProperties(&device_properties, 0);
    std::cout << "sharedMemPerBlock: " << device_properties.sharedMemPerBlock << std::endl;


/*
// Arguments to the partitioning function.
//
// Note that the struct's layout must be kept in sync with its counterpart in
// Rust.
struct RadixPartitionArgs {
  // Inputs
  const void *const __restrict__ join_attr_data;
  const void *const __restrict__ payload_attr_data;
  std::size_t const data_length;
  uint32_t const padding_length;
  uint32_t const radix_bits;
  uint32_t const ignore_bits;
  const unsigned long long *const __restrict__ partition_offsets;

  // State
  uint32_t *const __restrict__ tmp_partition_offsets;
  char *const __restrict__ l2_cache_buffers;
  char *const __restrict__ device_memory_buffers;
  uint64_t const device_memory_buffer_bytes;

  // Outputs
  void *const __restrict__ partitioned_relation;
};
*/

    RadixPartitionArgs args {
        nullptr,
        nullptr,
        0,
        0,
        22,
        8,
        nullptr
    };

    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes);*/

    //gpu_chunked_laswwc_radix_partition<<<1, 64>>>(args, );

    gpu_chunked_laswwc_radix_partition_int32_int32<<<1, 64>>>(args, device_properties.sharedMemPerBlock);
    hipDeviceSynchronize();

#if 0
    hipStream_t partition_stream, join_stream;
    hipStreamCreate(partition_stream);
    hipStreamCreate(join_stream);


    partition_kernel<<<1, 64, 0, partition_stream>>>();
    join_kernel<<<1, 64, 0, join_stream>>>();
    hipDeviceSynchronize();

#endif
    hipDeviceReset();

    return 0;
}
