#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <chrono>
#include <memory>

#include <cub/util_debug.cuh>

#include "cuda_utils.cuh"
#include "cuda_allocator.hpp"
#include "numa_allocator.hpp"
#include "mmap_allocator.hpp"
#include "indexes.cuh"
#include "device_array.hpp"

#include "index_lookup_config.cuh"
#include "index_lookup_common.cuh"

#include "gpu_prefix_sum.hpp"
#include "partitioned_relation.hpp"
#include "utils.hpp"

#undef GPU_CACHE_LINE_SIZE
#include <numa-gpu/sql-ops/include/gpu_radix_partition.h>
#include <numa-gpu/sql-ops/cudautils/gpu_common.cu>
#include <numa-gpu/sql-ops/cudautils/radix_partition.cu>


static const int num_streams = 2;
static const int block_size = 128;// 64;
static const int grid_size = 1;//1;
static const uint32_t radix_bits = 6;// 10;
static const uint32_t ignore_bits = 0;//3;

template<class T> using device_allocator_t = cuda_allocator<T, cuda_allocation_type::device>;
template<class T> using device_index_allocator_t = cuda_allocator<T, cuda_allocation_type::zero_copy>;





#if 0
const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    return 0;
}
#endif


__global__ void partition_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes)*/

}

__global__ void join_kernel(float *x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    }
}

void dump_offsets(const partition_offsets& offsets) {
    auto h_offsets = offsets.offsets.to_host_accessible();
    std::cout << stringify(h_offsets.data(), h_offsets.data() + h_offsets.size()) << std::endl;
    auto h_local_offsets = offsets.local_offsets.to_host_accessible();
    std::cout << stringify(h_local_offsets.data(), h_local_offsets.data() + h_local_offsets.size()) << std::endl;
}

struct stream_state {
    hipStream_t stream;

    size_t num_lookups;

    device_array_wrapper<int32_t> d_payloads;
    device_array_wrapper<index_key_t> d_dst_partition_attr;
    device_array_wrapper<int32_t> d_dst_payload_attrs;
    device_array_wrapper<value_t> d_dst_tids;

    device_array_wrapper<ScanState<unsigned long long>> d_prefix_scan_state;

    partition_offsets partition_offsets_inst;
    partitioned_relation<Tuple<index_key_t, int32_t>> partitioned_relation_inst;

    std::unique_ptr<PrefixSumArgs> prefix_sum_and_copy_args;
    std::unique_ptr<RadixPartitionArgs> radix_partition_args;
};

std::unique_ptr<stream_state> create_stream_state(const index_key_t* d_lookup_keys, size_t num_lookups) {
    auto state = std::make_unique<stream_state>();
//printf("num_lookups: %lu\n", num_lookups);
    CubDebugExit(hipStreamCreate(&state->stream));
/*
auto wrapper = device_array_wrapper<index_key_t>::create_reference_only(const_cast<index_key_t*>(d_lookup_keys), num_lookups);
auto r = wrapper.to_host_accessible();
*/

std::vector<index_key_t> tmp;
tmp.resize(num_lookups);
hipMemcpy(tmp.data(), d_lookup_keys, num_lookups*sizeof(index_key_t), hipMemcpyDeviceToHost);
std::cout << "input:" << stringify(tmp.begin(), tmp.end()) << std::endl;


    state->num_lookups = num_lookups;

    // dummy payloads
    state->d_payloads = create_device_array<int32_t>(num_lookups);

    // allocate output arrays
    state->d_dst_partition_attr = create_device_array<index_key_t>(num_lookups);
    state->d_dst_payload_attrs = create_device_array<int32_t>(num_lookups);
    state->d_dst_tids = create_device_array<value_t>(num_lookups);

    // see: device_exclusive_prefix_sum_initialize
    const auto prefix_scan_state_len = gpu_prefix_sum::state_size(grid_size, block_size);
    state->d_prefix_scan_state = create_device_array<ScanState<unsigned long long>>(prefix_scan_state_len);

    device_allocator_t<int> device_allocator;
    state->partition_offsets_inst = partition_offsets(grid_size, radix_bits, device_allocator);
    state->partitioned_relation_inst = partitioned_relation<Tuple<index_key_t, int32_t>>(num_lookups, grid_size, radix_bits, device_allocator);

    state->prefix_sum_and_copy_args = std::unique_ptr<PrefixSumArgs>(new PrefixSumArgs {
        // Inputs
        d_lookup_keys,
        num_lookups,
        0, // not used
        state->partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
        // State
        state->d_prefix_scan_state.data(),
        state->partition_offsets_inst.local_offsets.data(),
        // Outputs
        state->partition_offsets_inst.offsets.data()
    });

    state->radix_partition_args = std::unique_ptr<RadixPartitionArgs>(new RadixPartitionArgs {
        // Inputs
        d_lookup_keys,
        state->d_payloads.data(),
        num_lookups,
        state->partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
//        offsets.local_offsets.data(),
        state->partition_offsets_inst.offsets.data(),
        // State
        nullptr,
        nullptr,
        nullptr,
        0,
        // Outputs
        state->partitioned_relation_inst.relation.data()
    });

    return state;
}

template<class IndexStructureType>
__global__ void lookup_kernel(const IndexStructureType index_structure, unsigned n, const Tuple<index_key_t, int32_t>* __restrict__ relation, value_t* __restrict__ tids) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int i = index;
    uint32_t active_lanes = __ballot_sync(FULL_MASK, i < n);
    while (active_lanes) {
        bool active = i < n;
printf("lookup %u\n", relation[i].key);
        auto tid = index_structure.cooperative_lookup(active, relation[i].key);
        if (active) {
            tids[i] = tid;
            printf("tid %lu\n", tid);
        }

        i += stride;
        active_lanes = __ballot_sync(FULL_MASK, i < n);
    }
}


template<class K, class V>
std::string tmpl_to_string(const Tuple<K, V>& tuple) {
    return std::to_string(tuple.key);
}


template<class IndexStructureType>
void run_on_stream(stream_state& state, IndexStructureType& index_structure, const hipDeviceProp_t& device_properties) {
    const auto required_shared_mem_bytes = ((block_size + (block_size >> LOG2_NUM_BANKS)) + gpu_prefix_sum::fanout(radix_bits)) * sizeof(uint64_t);
    printf("required_shared_mem_bytes %lu\n", required_shared_mem_bytes);

    assert(required_shared_mem_bytes <= device_properties.sharedMemPerBlock);

    // prepare kernel arguments
    void* args[1];
    args[0] = state.prefix_sum_and_copy_args.get();

    CubDebugExit(hipLaunchCooperativeKernel(
        (void*)gpu_contiguous_prefix_sum_int32,
        dim3(grid_size),
        dim3(block_size),
        args,
        required_shared_mem_bytes,
        state.stream
    ));

    const auto required_shared_mem_bytes_2 = gpu_prefix_sum::fanout(radix_bits) * sizeof(uint32_t);

    //gpu_chunked_radix_partition_int32_int32<<<grid_size, block_size, required_shared_mem_bytes_2, state.stream>>>(*state.radix_partition_args);
    gpu_chunked_laswwc_radix_partition_int32_int32<<<grid_size, block_size, device_properties.sharedMemPerBlock, state.stream>>>(*state.radix_partition_args, device_properties.sharedMemPerBlock);



hipDeviceSynchronize();
auto r = state.partitioned_relation_inst.relation.to_host_accessible();
std::cout << "result:" << stringify(r.data(), r.data() + state.num_lookups) << std::endl;
return;
    lookup_kernel<<<grid_size, block_size, 4*1024, state.stream>>>(index_structure.device_index, state.num_lookups, state.partitioned_relation_inst.relation.data(), state.d_dst_tids.data());
}

int main(int argc, char** argv) {
    double zipf_factor = 1.25;
    auto num_elements = default_num_elements;
    size_t num_lookups = 1024;// default_num_lookups;
    if (argc > 1) {
        std::string::size_type sz;
        num_elements = std::stod(argv[1], &sz);
    }
    std::cout << "index size: " << num_elements << std::endl;

    // generate datasets
    std::vector<index_key_t, host_allocator_t<index_key_t>> indexed, lookup_keys;
    indexed.resize(num_elements);
    lookup_keys.resize(num_lookups);
    generate_datasets<index_key_t, index_type>(dataset_type::dense, max_bits, indexed, lookup_pattern_type::uniform, zipf_factor, lookup_keys);
std::cout << stringify(lookup_keys.begin(), lookup_keys.end());
//return 0;
    // create gpu accessible vectors
    indexed_allocator_t indexed_allocator;
    auto d_indexed = create_device_array_from(indexed, indexed_allocator);
    lookup_keys_allocator_t lookup_keys_allocator;
    auto d_lookup_keys = create_device_array_from(lookup_keys, lookup_keys_allocator);
    auto index = build_index<index_key_t, index_type>(indexed, d_indexed.data());
/*
auto r = d_lookup_keys.to_host_accessible();
std::cout << "result:" << stringify(r.data(), r.data() + num_lookups) << std::endl;
return 0;
*/
    // fetch device properties
    hipDeviceProp_t device_properties;
    CubDebugExit(hipGetDeviceProperties(&device_properties, 0));
    std::cout << "sharedMemPerBlock: " << device_properties.sharedMemPerBlock << std::endl;

#if 0
    device_allocator_t<int> device_allocator;

    // dummy payloads
    auto d_payloads = create_device_array<int32_t>(num_lookups);

    // allocate output arrays
    auto dst_partition_attr = create_device_array<index_key_t>(num_lookups);
    auto dst_payload_attrs = create_device_array<int32_t>(num_lookups);

    //ScanState<unsigned long long>* prefix_scan_state; // see: device_exclusive_prefix_sum_initialize
    const auto prefix_scan_state_len = gpu_prefix_sum::state_size(grid_size, block_size);
    auto prefix_scan_state = create_device_array<ScanState<unsigned long long>>(prefix_scan_state_len);

    partition_offsets offsets(grid_size, radix_bits, device_allocator);
    //partitioned_relation<index_key_t> partitioned_relation_inst(num_lookups, grid_size, radix_bits, device_allocator);
    partitioned_relation<Tuple<int32_t, int32_t>> partitioned_relation_inst(num_lookups, grid_size, radix_bits, device_allocator);

    PrefixSumArgs prefix_sum_and_copy_args {
        // Inputs
        d_lookup_keys.data(),
        num_lookups,
        0, // not used
        partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
        // State
        prefix_scan_state.data(),
        offsets.local_offsets.data(),
        // Outputs
        offsets.offsets.data()
    };

    //__host__ ​hipError_t hipLaunchCooperativeKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, void** args, size_t sharedMem, hipStream_t stream ) 
    /*
    template <typename K, typename V>
    __device__ void gpu_contiguous_prefix_sum_and_copy_with_payload(args)
    */
    hipStream_t scan_stream;
    CubDebugExit(hipStreamCreate(&scan_stream));

    const auto required_shared_mem_bytes = ((block_size + (block_size >> LOG2_NUM_BANKS)) + gpu_prefix_sum::fanout(radix_bits)) * sizeof(uint64_t);
    printf("required_shared_mem_bytes %lu\n", required_shared_mem_bytes);
    //const void* func = &gpu_contiguous_prefix_sum_and_copy_with_payload<int, int>;
    assert(required_shared_mem_bytes <= device_properties.sharedMemPerBlock);

    // prepare kernel arguments
    //auto d_prefix_sum_and_copy_args = create_device_array_from(reinterpret_cast<const uint8_t*>(&prefix_sum_and_copy_args), sizeof(prefix_sum_and_copy_args));
    void* args[1];
    args[0] = &prefix_sum_and_copy_args;// d_prefix_sum_and_copy_args.data();

    CubDebugExit(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(
        //func),
//        (void*)gpu_contiguous_prefix_sum_and_copy_with_payload_int32_int32,
        //(void*)gpu_chunked_prefix_sum_int32,
        (void*)gpu_contiguous_prefix_sum_int32,
        dim3(grid_size),
        dim3(block_size),
        args,
        required_shared_mem_bytes,
        scan_stream
    ));
    hipDeviceSynchronize();
    printf("gpu_contiguous_prefix_sum_and_copy_with_payload_int32_int32 done\n");

dump_offsets(offsets);
/*
// Arguments to the partitioning function.
//
// Note that the struct's layout must be kept in sync with its counterpart in
// Rust.
struct RadixPartitionArgs {
  // Inputs
  const void *const __restrict__ join_attr_data;
  const void *const __restrict__ payload_attr_data;
  std::size_t const data_length;
  uint32_t const padding_length;
  uint32_t const radix_bits;
  uint32_t const ignore_bits;
  const unsigned long long *const __restrict__ partition_offsets;

  // State
  uint32_t *const __restrict__ tmp_partition_offsets;
  char *const __restrict__ l2_cache_buffers;
  char *const __restrict__ device_memory_buffers;
  uint64_t const device_memory_buffer_bytes;

  // Outputs
  void *const __restrict__ partitioned_relation;
};
*/

    RadixPartitionArgs radix_partition_args {
        // Inputs
        //dst_partition_attr.data(),
        //dst_payload_attrs.data(),
        d_lookup_keys.data(),
        d_payloads.data(),
        num_lookups,
        partitioned_relation_inst.padding_length(),
        radix_bits,
        ignore_bits,
//        offsets.local_offsets.data(),
        offsets.offsets.data(),
        // State
        nullptr,
        nullptr,
        nullptr,
        0,
        // Outputs
        partitioned_relation_inst.relation.data()
    };

    const auto required_shared_mem_bytes_2 = gpu_prefix_sum::fanout(radix_bits) * sizeof(uint32_t);

    /*
    template <typename K, typename V>
    __device__ void gpu_chunked_laswwc_radix_partition(RadixPartitionArgs &args, uint32_t shared_mem_bytes);
    */

    //gpu_chunked_laswwc_radix_partition<<<1, 64>>>(args, );

//gpu_chunked_laswwc_radix_partition_int32_int32<<<grid_size, block_size, device_properties.sharedMemPerBlock, scan_stream>>>(radix_partition_args, device_properties.sharedMemPerBlock);

    gpu_chunked_radix_partition_int32_int32<<<grid_size, block_size, required_shared_mem_bytes_2, scan_stream>>>(radix_partition_args);
    hipDeviceSynchronize();


    printf("gpu_chunked_laswwc_radix_partition_int32_int32 done\n");

#if 0
    hipStream_t partition_stream, join_stream;
    hipStreamCreate(partition_stream);
    hipStreamCreate(join_stream);


    partition_kernel<<<1, 64, 0, partition_stream>>>();
    join_kernel<<<1, 64, 0, join_stream>>>();
    hipDeviceSynchronize();

#endif
    hipDeviceReset();
#endif




    size_t remaining = num_lookups;
    size_t max_stream_portion = num_lookups / num_streams;
    const index_key_t* d_stream_lookup_keys = d_lookup_keys.data();
/*
auto r = d_lookup_keys.to_host_accessible();
std::cout << "input:" << stringify(r.data(), r.data() + num_lookups) << std::endl;
*/
    //printf("estimated partition size: %lu\n", partition_size);

    std::vector<std::unique_ptr<stream_state>> stream_states;

    // create streams
    for (unsigned i = 0; i < num_streams; ++i) {
        size_t stream_portion = std::min(remaining, max_stream_portion);
        remaining -= stream_portion;
printf("stream portion: %lu\n", stream_portion);
        auto state = create_stream_state(d_stream_lookup_keys, stream_portion);
        stream_states.push_back(std::move(state));

        d_stream_lookup_keys += stream_portion;
    }

    for (const auto& state : stream_states) {
        run_on_stream(*state, *index, device_properties);
    }
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}
